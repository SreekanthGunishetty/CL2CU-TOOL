#include "hip/hip_runtime.h"
__global__ void dot_product( float4* a_vec,  float4* b_vec,  float* output, __local float4* partial_dot) {

int gid = blockIdx.x * blockDim.x + threadIdx.x;
int lid = threadIdx.x;
int group_size = blockDim.x;

partial_dot[lid] = a_vec[gid] * b_vec[gid];
__syncthreads()

for(int i = group_size/2; i>0; i >>= 1) {
if(lid < i) {
partial_dot[lid] += partial_dot[lid + i];
}
__syncthreads()
}

if(lid == 0) {
output[blockIdx.x] = dot(partial_dot[0], (float4)(1.0f));
}
}


#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>




#define _CRT_SECURE_NO_WARNINGS
#define PROGRAM_FILE "dot_product.cl"

// 2^18
#define VEC_SIZE 262144

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <time.h>

#ifdef MAC

#else

#endif


int main() {






size_t max_local_size, global_size;

struct timeval start, finish;
cl_event prof_event;
cl_ulong time_start, time_end;

float a_vec[VEC_SIZE];
float b_vec[VEC_SIZE];
float dot_output, dot_check, result;
float *output_vec;
void* a_buffer, b_buffer, output_buffer;

srand((unsigned int)time(0));
for(i=0; i<VEC_SIZE; i++) {
a_vec[i] = (float)rand()/RAND_MAX;
}
srand((unsigned int)time(0));
for(i=0; i<VEC_SIZE; i++) {
b_vec[i] = (float)rand()/RAND_MAX;
}

dot_check = 0.0f;
gettimeofday(&start, NULL);
for(i=0; i<VEC_SIZE; i++) {
dot_check += a_vec[i] * b_vec[i];
}
gettimeofday(&finish, NULL);
printf("On the host, the dot product took %u.%06u seconds.\n",(unsigned int)(finish.tv_sec - start.tv_sec),(unsigned int)(finish.tv_usec - start.tv_usec));











num_groups = (VEC_SIZE/4)/max_local_size;
output_vec = (float*) malloc(num_groups * sizeof(float));




















/*CU2CL Unsupported -- Unsupported CUDA call: clCreateBuffer*/
hipMalloc((void**)&a_buffer ,sizeof(a_vec));
/*CU2CL Unsupported -- Unsupported CUDA call: clCreateBuffer*/
hipMalloc((void**)&b_buffer , sizeof(b_vec));
/*CU2CL Unsupported -- Unsupported CUDA call: clCreateBuffer*/
hipMalloc((void**)&output_buffer ,num_groups * sizeof(float));









global_size = VEC_SIZE/4;
dot_product<<<global_size,max_local_size>>>(a_buffer,b_buffer,output_buffer,NULL);

hipMemcpy(output_vec, output_buffer, num_groups * sizeof(float),hipMemcpyDeviceToHost);

/*CU2CL Unsupported -- Unsupported CUDA call: clGetEventProfilingInfo*/
clGetEventProfilingInfo(prof_event, CL_PROFILING_COMMAND_START,sizeof(time_start), &time_start, NULL);
/*CU2CL Unsupported -- Unsupported CUDA call: clGetEventProfilingInfo*/
clGetEventProfilingInfo(prof_event, CL_PROFILING_COMMAND_END, sizeof(time_end), &time_end, NULL);
printf("On the device, the dot product kernel completed in %lu ns.\n",(time_end - time_start));

dot_output = 0.0f;
for(i=0; i<num_groups; i++)
dot_output += output_vec[i];

result = fabs(dot_output - dot_check);
if(result > 10.0f)
printf("Dot product failed.\n");
else
printf("Dot product succeeded.\n");

free(output_vec);
hipFree(a_buffer);
hipFree(b_buffer);
hipFree(output_buffer);




return 0;
}

