#include "hip/hip_runtime.h"
__global__ void mat_mul( int *A,  int *B,  int *C)
{
int n = 3, m = 3, l = 3;
int idx = blockIdx.x * blockDim.x + threadIdx.x;
int idy = blockIdx.y * blockDim.y + threadIdx.y;
if (idx < 3 && idy < 3) {
int i, sum = 0;
for (i = 0 ; i < n ; ++i) {
sum += A[idy*n+i] * B[idx+i*n];
}
C[idx+idy*n] = sum;
}
}


#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>




#include <stdio.h>

#include <stdlib.h>
#include <fcntl.h>

#define BUF_SIZE 2048
#define ERROR -1

#define END 0

char *KernelSource = "";

void printMatrix(int *arr, int n, int m) {
int i, j;
for (i = 0 ; i < n ; ++i ) {
for (j = 0 ; j < m ; ++j ) {
printf("%d ", arr[i*n+j]);
}
printf("\n" );
}
printf("\n" );
}

int main(int argc, char const *argv[]) {








void* inputA, inputB, output;
dim3 local_item_size[2], global_item_size[2];
int len;
int res;
int n = 3;
int inp_len = sizeof(int)*n*n, op_len = sizeof(int)*n*n;
int matA[] = {1,2,5,4,2,5,5,2,6}, matB[] = {5,2,7,3,6,2,6,2,6};
int matC[9];

printMatrix(matA,n,n);
printMatrix(matB,n,n);


//


//






/*CU2CL Unsupported -- Unsupported CUDA call: clCreateBuffer*/
hipMalloc((void**)&inputA ,inp_len);
/*CU2CL Unsupported -- Unsupported CUDA call: clCreateBuffer*/
hipMalloc((void**)&inputB ,inp_len);
/*CU2CL Unsupported -- Unsupported CUDA call: clCreateBuffer*/
hipMalloc((void**)&output ,op_len);

hipMemcpy(inputA, matA, inp_len,hipMemcpyHostToDevice);
hipMemcpy(inputB, matB, inp_len,hipMemcpyHostToDevice);


















//






global_item_size.x = 3;
global_item_size.y = 3;

mat_mul<<<global_item_size,NULL>>>(inputA,inputB,output);

hipMemcpy(matC, output, op_len,hipMemcpyDeviceToHost);

printMatrix(matC,n,n);

return 0;
}

