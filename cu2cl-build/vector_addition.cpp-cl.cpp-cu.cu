#include "hip/hip_runtime.h"
__global__ void vecAdd(   double *a,  double *b,  double *c, const unsigned int n)
{
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id < n)
c[id] = a[id] + b[id];
}


#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
int main( int argc, char* argv[] )
{
unsigned int n = 100;
double *h_a;
double *h_b;
double *h_c;
void* d_a;
void* d_b;
void* d_c;
size_t bytes = n*sizeof(double);
h_a = (double*)malloc(bytes);
h_b = (double*)malloc(bytes);
h_c = (double*)malloc(bytes);
int i;
for( i = 0; i < n; i++ )
{
h_a[i] = i;
h_b[i] = n-i;
}
size_t globalSize, localSize;
localSize = 64;
globalSize = ceil(n/(float)localSize)*localSize;
/*CU2CL Unsupported -- Unsupported CUDA call: clCreateBuffer*/
hipMalloc((void**)&d_a , bytes);
/*CU2CL Unsupported -- Unsupported CUDA call: clCreateBuffer*/
hipMalloc((void**)&d_b , bytes);
/*CU2CL Unsupported -- Unsupported CUDA call: clCreateBuffer*/
hipMalloc((void**)&d_c , bytes);
hipMemcpy(d_a, h_a, bytes,hipMemcpyHostToDevice);
hipMemcpy(d_b, h_b, bytes,hipMemcpyHostToDevice);
vecAdd<<<globalSize,localSize>>>(d_a,d_b,d_c,n);
hipDeviceSynchronize();
hipMemcpy(h_c, d_c, bytes,hipMemcpyDeviceToHost);
double sum = 0;
for(i=0; i<n; i++)
printf("%lf\n",h_c[i]);
hipFree(d_a);
hipFree(d_b);
hipFree(d_c);
free(h_a);
free(h_b);
free(h_c);
return 0;
}
