#include "hip/hip_runtime.h"

inline unsigned char get_pix( unsigned char* image, int width, int height, int y, int x) {
if (x < 0 || x >= width)
return 0;
if (y < 0 || y >= height)
return 0;
return image[y*width + x];
}

__global__ void sobel( unsigned char* imageIn,  unsigned char* imageOut,
const int width, const int height) {

int i = blockIdx.y * blockDim.y + threadIdx.y;
int j = blockIdx.x * blockDim.x + threadIdx.x;

int li = threadIdx.y;
int lj = threadIdx.x;

const int l_width = 16;
const int l_height = 16;

 unsigned char l_pixels[18][18];


char skip_WI = 0;
if (j >= width || i >= height) {
skip_WI = 1;
}

if (skip_WI == 0) {
//UPPER AND LOWER (LOCAL) EDGE
if (li == 0) {
l_pixels[li][lj+1] = get_pix(imageIn, width, height, i-1, j);
} else if (li == l_height-1) {
l_pixels[li+2][lj+1] = get_pix(imageIn, width, height, i+1, j);
}

// LEFT AND RIGHT (LOCAL) EDGE
if (lj == 0) {
l_pixels[li+1][lj] = get_pix(imageIn, width, height, i, j-1);
} else if (lj == l_width-1) {
l_pixels[li+1][lj+2] = get_pix(imageIn, width, height, i, j+1);
}

// (LOCAL) CORNERS
if (li == 0 && lj == 0) {
// UPPER LEFT
l_pixels[li][lj] = get_pix(imageIn, width, height, i-1, j-1);
} else if (li == l_height-1 && lj == 0) {
// LOWER LEFT
l_pixels[li+2][lj] = get_pix(imageIn, width, height, i+1, j-1);
} else if (li == 0 && lj == l_width-1) {
// UPPER RIGHT
l_pixels[li][lj+2] = get_pix(imageIn, width, height, i-1, j+1);
} else if (li == l_height-1 && lj == l_width-1) {
// LOWER RIGHT
l_pixels[li+2][lj+2] = get_pix(imageIn, width, height, i+1, j+1);
}

// MAIN PIXEL
l_pixels[li+1][lj+1] = get_pix(imageIn, width, height, i, j);
}

__syncthreads()

if (skip_WI == 1)
return;

int Gx, Gy;

Gx = -1 * l_pixels[li][lj] - 2 * l_pixels[li][lj+1] - 1 * l_pixels[li][lj+2] +
1 * l_pixels[li+2][lj] + 2 * l_pixels[li+2][lj+1] +  1 * l_pixels[li+2][lj+2];

Gy = -1 * l_pixels[li][lj] - 2 * l_pixels[li+1][lj] - 1 * l_pixels[li+2][lj] +
1 * l_pixels[li][lj+2] + 2 * l_pixels[li+1][lj+2] + 1 * l_pixels[li+2][lj+2];

int tmpPix = sqrt((float)(Gx * Gx + Gy * Gy));

if (tmpPix > 255)
imageOut[i*width + j] = 255;
else
imageOut[i*width + j] = tmpPix;

}


#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "FreeImage.h-cl.h"
#define MAX_SOURCE_SIZE 10000
/*char* readKernel(const char* file) {
exit(1);
}
}*/
int main(void)
{
unsigned char *slikaInput;
unsigned char *slikaOutput;
FIBITMAP *imageBitmap = FreeImage_Load(FIF_PNG, "lena_sp_noise.png", 0);
FIBITMAP *imageBitmapGrey = FreeImage_ConvertToGreyscale(imageBitmap);
int width = FreeImage_GetWidth(imageBitmapGrey);
int height = FreeImage_GetHeight(imageBitmapGrey);
unsigned char *imageIn = (unsigned char*)malloc(height*width * sizeof(unsigned char));
unsigned char *imageOut = (unsigned char*)malloc(height*width * sizeof(unsigned char));
FreeImage_ConvertToRawBits(imageIn, imageBitmapGrey, width, 8, 0xFF, 0xFF, 0xFF, TRUE);
FreeImage_Unload(imageBitmapGrey);
FreeImage_Unload(imageBitmap);
dim3 global_item_size;
dim3 local_item_size;
local_item_size.x = 16;
local_item_size.y = 16;
global_item_size.x = width;
while (global_item_size.x % 16 != 0) {
global_item_size[0]++;
}
global_item_size.y = height;
while (global_item_size.y % 16 != 0) {
global_item_size[1]++;
}
printf("Global worksize: %ld x %ld\n", (long) global_item_size[0], (long) global_item_size[1]);
printf("Local worksize: %ld x %ld\n", (long) local_item_size[0], (long) local_item_size[1]);
size_t atom_buffer_size = height * width * sizeof(unsigned char);
/*CU2CL Unsupported -- Unsupported CUDA call: clCreateBuffer*/
void* in_img_cl;
hipMalloc((void**)&in_img_cl,atom_buffer_size);
/*CU2CL Unsupported -- Unsupported CUDA call: clCreateBuffer*/
void* out_img_cl;
hipMalloc((void**)&out_img_cl, atom_buffer_size);
hipMemcpy(in_img_cl, imageIn, sizeof(unsigned char) * height * width,hipMemcpyHostToDevice);
sobel<<<global_item_size,local_item_size>>>(in_img_cl,out_img_cl,width,height);
hipDeviceSynchronize();
hipMemcpy(imageOut, out_img_cl, atom_buffer_size,hipMemcpyDeviceToHost);
FIBITMAP *imageOutBitmap = FreeImage_ConvertFromRawBits(imageOut, width, height, width, 8, 0xFF, 0xFF, 0xFF, TRUE);
FreeImage_Save(FIF_PNG, imageOutBitmap, "result.png", 0);
FreeImage_Unload(imageOutBitmap);
hipDeviceSynchronize();
hipFree(in_img_cl);
hipFree(out_img_cl);
free(global_work_size);
free(local_work_size);
free(imageIn);
free(imageOut);
return 0;
}
