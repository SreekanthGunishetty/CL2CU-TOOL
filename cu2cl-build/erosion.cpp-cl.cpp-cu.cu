#include "hip/hip_runtime.h"
__global__ erosion( unsigned char* input,  unsigned char* output, const unsigned int width, const unsigned int height)
{
int gid_x = blockIdx.x * blockDim.x + threadIdx.x;
int gid_y = blockIdx.y * blockDim.y + threadIdx.y;


int id = (gid_y * width) + gid_x;

if (gid_y < height && gid_x < width)
{
if (gid_x == 0 || gid_x == width - 1 || gid_y == 0 || gid_y == height - 1)
output[id] = input[id];
else
{
int window[3];

int id0 = ((gid_y - 1) * width) + (gid_x);
window[0] = input[id0];
int id1 = ((gid_y) * width) + gid_x;
window[1] = input[id1];
int id2 = ((gid_y + 1) * width) + (gid_x);
window[2] = input[id2];

int min = window[0];
for (a = 0; a < 3; a++)
{
if (window[a] < min)
min = window[a];
}
output[id] = min;
}
}
}


#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>




#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/core/types_c.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>


using namespace cv;
using namespace std;


int main()
{

Mat mat = imread("lena_sp_noise.png", CV_LOAD_IMAGE_GRAYSCALE);
unsigned int width = mat.cols;
unsigned int height = mat.rows;
uchar* input = new uchar[(height * width)];
if (mat.isContinuous())
input = mat.data;

uchar* output = new uchar[(width * height)];
for (int i = 0; i < width * height; ++i)
output[i] = input[i];

void* i_mem, o_mem;







size_t bytes = (width * height) * sizeof(char);



























/*CU2CL Unsupported -- Unsupported CUDA call: clCreateBuffer*/
hipMalloc((void**)&i_mem , bytes);
/*CU2CL Unsupported -- Unsupported CUDA call: clCreateBuffer*/
hipMalloc((void**)&o_mem , bytes);
hipMemcpy(i_mem, input, bytes,hipMemcpyHostToDevice);
hipDeviceSynchronize();






size_t globalSize = width * height;
erosion<<<&globalSize,NULL>>>(i_mem,o_mem,width,height);
hipDeviceSynchronize();

hipMemcpy(output, o_mem, bytes,hipMemcpyDeviceToHost);






free(i_mem);
free(o_mem);



Mat finImg = Mat(height, width, CV_8UC1, output);
imwrite("eroded.png", finImg);

delete[] input;
delete[] output;
return 0;
}


