#include "hip/hip_runtime.h"
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp> 
#include <iostream> 
#include <string> 
#include <stdio.h> 
#include <hip/hip_runtime.h> 
#include "hip/hip_runtime.h" 
#define BLOCK_SIZE 16 
#define FILTER_WIDTH 3 
#define FILTER_HEIGHT 3

using namespace std; 
extern "C" void medianFilter_GPU_wrapper(const cv::Mat& input, cv::Mat& output);
int main( int argc, char** argv ) 
{
   string image_name = "lena_sp_noise";
   string input_file = image_name+".png";
   //string output_file_cpu = image_name+"_cpu.png";
   string output_file_gpu = image_name+"_gpu.png";
   cv::Mat srcImage = cv::imread(input_file ,CV_LOAD_IMAGE_UNCHANGED);
   if(srcImage.empty())
   {
      std::cout<<"Image Not Found: "<< input_file << std::endl;
      return -1;
   }
   cout <<"\ninput image size: "<<srcImage.cols<<" "<<srcImage.rows<<" "<<srcImage.channels()<<"\n";
    
     
   cv::Mat dstImage (srcImage.size(), srcImage.type());
    
   medianFilter_GPU_wrapper(srcImage, dstImage);
   
   imwrite(output_file_gpu, dstImage);
   
   //medianFilter_CPU(srcImage, dstImage);
   
   //imwrite(output_file_cpu, dstImage);
       
   return 0;
}
__device__ void sort(unsigned char* filterVector) 
{
	 for (int i = 0; i < FILTER_WIDTH*FILTER_HEIGHT; i++)
	 {
	    for (int j = i + 1; j < FILTER_WIDTH*FILTER_HEIGHT; j++)
	    {
			if (filterVector[i] > filterVector[j])
			{
					  //Swap the variables
				  unsigned char tmp = filterVector[i];
				  filterVector[i] = filterVector[j];
				  filterVector[j] = tmp;
			}
            }
         }
}
__global__ void medianFilter(unsigned char *srcImage, unsigned char *dstImage, unsigned int width, unsigned int height, int channel) {
   int x = blockIdx.x*blockDim.x + threadIdx.x;
   int y = blockIdx.y*blockDim.y + threadIdx.y;
  
   if((x>=FILTER_WIDTH/2) && (x<(width-FILTER_WIDTH/2)) && (y>=FILTER_HEIGHT/2) && (y<(height-FILTER_HEIGHT/2)))
   {
      for(int c=0 ; c<channel ; c++)
      {
         unsigned char filterVector[FILTER_WIDTH*FILTER_HEIGHT];
         // Loop inside the filter to average pixel values
         for(int ky=-FILTER_HEIGHT/2; ky<=FILTER_HEIGHT/2; ky++)
		 {
            for(int kx=-FILTER_WIDTH/2; kx<=FILTER_WIDTH/2; kx++)
			{
               filterVector[ky*FILTER_WIDTH+kx] = srcImage[((y+ky)*width + (x+kx))*channel+c];
            }
         }
          
         sort(filterVector);
         dstImage[(y*width+x)*channel+c] = filterVector[(FILTER_WIDTH*FILTER_HEIGHT)/2];
      }
   }
}
extern "C" void medianFilter_GPU_wrapper(const cv::Mat& input, cv::Mat& output) 
{
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        int channel = input.step/input.cols;
        const int inputSize = input.cols * input.rows * channel;
        const int outputSize = output.cols * output.rows * channel;
        unsigned char *d_input, *d_output;
        
        hipMalloc<unsigned char>(&d_input,inputSize);
        hipMalloc<unsigned char>(&d_output,outputSize);
        hipMemcpy(d_input,input.ptr(),inputSize,hipMemcpyHostToDevice);
        const dim3 block(BLOCK_SIZE,BLOCK_SIZE);
        const dim3 grid((output.cols + block.x - 1)/block.x, (output.rows + block.y - 1)/block.y);
        hipEventRecord(start);
        medianFilter<<<grid,block>>>(d_input, d_output, output.cols, output.rows, channel);
        hipEventRecord(stop);
        hipMemcpy(output.ptr(),d_output,outputSize,hipMemcpyDeviceToHost);
        hipFree(d_input);
        hipFree(d_output);
        hipEventSynchronize(stop);
        float milliseconds = 0;
          
        hipEventElapsedTime(&milliseconds, start, stop);
        cout<< "\nProcessing time on GPU (ms): " << milliseconds << "\n";
}
