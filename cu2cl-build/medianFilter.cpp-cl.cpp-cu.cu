#include "hip/hip_runtime.h"
__global__ void median_filter( unsigned char *inp,  unsigned char *op, const int height,const int width){

int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;
// printf("%d is width and %d is height\n",width,height);
int id = (x * width) + y;

if (y < width && x < height)
{
if (x == 0 || y == 0 || x == height - 1 || y == width - 1)
{
op[id] = inp[id];
}
else
{
int window[9];

int c = 0;
for (int k = 0; k < 3; k++)
{
window[c++] = inp[(x - 1) * width + y - 1 + k];
}
for (int k = 0; k < 3; k++)
{
window[c++] = inp[(x)*width + y - 1 + k];
}
for (int k = 0; k < 3; k++)
{
window[c++] = inp[(x + 1) * width + y - 1 + k];
}
for (int k = 1; k < 9; k++)
{
int key = window[k];
int l;
for (l = k - 1; l >= 0 && window[l] > key; l--)
{
window[l + 1] = window[l];
}
window[l + 1] = key;
}
// median
op[id] = window[4];
}
}
}


#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <iostream>
#include <time.h>
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/core/types_c.h>
using namespace std;
#define MAX_SOURCE_SIZE (0x100000)
// insertion sort is done, doesnt matter a lot which sort as it is just 9 elements, could also use the default sort in cpp
void Sort(int* window)
{
for (int i = 1; i < 9; i++)
{
int key = window[i];
int j;
for (j = i - 1; j >= 0 && window[j] > key; j--)
{
window[j + 1] = window[j];
}
window[j + 1] = key;
}
}
int main(void) {
cv::Mat image = cv::imread("input3.png", cv::IMREAD_GRAYSCALE);
if (image.empty())
{
cout << "File not present, check the input name.." << endl;
cin.get();
exit(1);
}
int w = image.cols;
int h = image.rows;
uchar* img = new uchar[w * h];
img = image.data;
/*CU2CL Unsupported -- Unsupported CUDA call: clCreateBuffer*/
void* a_mem_obj;
hipMalloc((void**)&a_mem_obj, w * h * sizeof(uchar));
/*CU2CL Unsupported -- Unsupported CUDA call: clCreateBuffer*/
void* c_mem_obj;
hipMalloc((void**)&c_mem_obj, w * h * sizeof(uchar));
hipMemcpy(a_mem_obj, img, w * h * sizeof(uchar),hipMemcpyHostToDevice);
hipMemcpy(c_mem_obj, img, w * h * sizeof(uchar),hipMemcpyHostToDevice);
dim3 global_item_size;
global_item_size.x = w;
global_item_size.y = h;
dim3 local_item_size;
local_item_size.x = 1;
local_item_size.y = 1;
clock_t start, end;
/*CU2CL Unsupported -- Unsupported CUDA call: clock*/
start = clock();
median_filter<<<global_item_size,local_item_size>>>(a_mem_obj,c_mem_obj,h,w);
hipDeviceSynchronize();
/*CU2CL Unsupported -- Unsupported CUDA call: clock*/
end = clock();
double time = ((double)end - (double)start) / CLOCKS_PER_SEC;
printf("Time taken is: %lf\n",time);
uchar* op = new uchar[w * h];
hipMemcpy(op, c_mem_obj, w * h * sizeof(uchar),hipMemcpyDeviceToHost);
hipDeviceSynchronize();
cv::Mat output = cv::Mat(h, w, CV_8UC1, op);
cv::imwrite("fromgpu3.jpg", output);
hipDeviceSynchronize();
hipFree(a_mem_obj);
hipFree(c_mem_obj);
return 0;
}
