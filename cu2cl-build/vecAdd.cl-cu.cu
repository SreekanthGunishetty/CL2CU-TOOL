#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>





#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <CL/opencl.h>


int main( int argc, char* argv[] )
{
unsigned int n = 100;

double *h_a;
double *h_b;
double *h_c;

cl_mem d_a;
cl_mem d_b;
cl_mem d_c;

cl_platform_id cpPlatform;        // OpenCL platform
cl_device_id device_id;           // device ID
cl_context context;               // context
cl_command_queue queue;           // command queue
cl_program program;               // program
cl_kernel kernel;                 // kernel

size_t bytes = n*sizeof(double);

h_a = (double*)malloc(bytes);
h_b = (double*)malloc(bytes);
h_c = (double*)malloc(bytes);

int i;
for( i = 0; i < n; i++ )
{
h_a[i] = i;
h_b[i] = n-i;
}

size_t globalSize, localSize;
//    cl_int err;

localSize = 64;

globalSize = ceil(n/(float)localSize)*localSize;

//;

//;

/*CU2CL Unsupported -- Unsupported CUDA call: clCreateContext*/
context = clCreateContext(0, 1, &device_id, NULL, NULL, NULL);

/*CU2CL Unsupported -- Unsupported CUDA call: clCreateCommandQueue*/
queue = clCreateCommandQueue(context, device_id, 0, NULL);
char* kernelSource;
FILE* kernelFile;
size_t kernelSize;
kernelFile = fopen("vecAdd.cl","rb");
fseek(kernelFile, 0, SEEK_END);
kernelSize = ftell(kernelFile);
rewind(kernelFile);
kernelSource = (char*)malloc(kernelSize +1);
kernelSource[kernelSize] = '\0';
fread(kernelSource, sizeof(char), kernelSize, kernelFile);
fclose(kernelFile);
/*CU2CL Unsupported -- Unsupported CUDA call: clCreateProgramWithSource*/
program = clCreateProgramWithSource(context, 1, (const char **)&kernelSource, &kernelSize, NULL);

//;

kernel = clCreateKernel();

/*CU2CL Unsupported -- Unsupported CUDA call: clCreateBuffer*/
d_a = clCreateBuffer(context, CL_MEM_READ_ONLY, bytes, NULL, NULL);
/*CU2CL Unsupported -- Unsupported CUDA call: clCreateBuffer*/
d_b = clCreateBuffer(context, CL_MEM_READ_ONLY, bytes, NULL, NULL);
/*CU2CL Unsupported -- Unsupported CUDA call: clCreateBuffer*/
d_c = clCreateBuffer(context, CL_MEM_WRITE_ONLY, bytes, NULL, NULL);

hipMemcpy(d_a, h_a, bytes,hipMemcpyHostToDevice);
hipMemcpy(d_b, h_b, bytes,hipMemcpyHostToDevice);

//;
//;
//;
//;

vecAdd<<<globalSize,localSize>>>(d_a,d_b,d_c,n);

hipDeviceSynchronize();

hipMemcpy(h_c, d_c, bytes,hipMemcpyDeviceToHost);

double sum = 0;
for(i=0; i<n; i++)
printf("%lf\n",h_c[i]);

hipFree(d_a);
hipFree(d_b);
hipFree(d_c);
//;
//;
//;
//;

free(h_a);
free(h_b);
free(h_c);

return 0;
}


