#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>

const int N = 1024;		// The matrix used will be of size (N,N)
const int K = 32;		// This is for the tile size (tile size is (K,K))

// To compare the outputs obtained from the GPU with that of the CPU
int compare_matrices(float* gpu, float* ref)
{
	for (int i = 0; i < N * N; i++)
		if (gpu[i] != ref[i])
			return 1;
	return 0;
}

// To fill a matrix with sequential numbers in the range 0..N-1
void fill_matrix(float* mat)
{
	for (int j = 0; j < N * N; j++)
		mat[j] = (float)j;
}

// CPU code for the serial execution of matrix transpose
void transpose_CPU(float in[], float out[])
{
	for (int j = 0; j < N; j++)
		for (int i = 0; i < N; i++)
			out[j + i * N] = in[i + j * N]; 
}

// Serial function to be launched on a single thread of the GPU
__global__ void transpose_serial(float in[], float out[])
{
	for (int j = 0; j < N; j++)
		for (int i = 0; i < N; i++)
			out[j + i * N] = in[i + j * N]; 
}

// One thread per row of output matrix (unrolling by N)
__global__ void transpose_parallel_per_row(float in[], float out[])
{
	int i = threadIdx.x;

	for (int j = 0; j < N; j++)
		out[j + i * N] = in[i + j * N]; 
}

// One thread per element, in KxK threadblocks, thread (x,y) in grid writes element (i,j) of output matrix 
__global__ void transpose_parallel_per_element(float in[], float out[])
{
	int i = blockIdx.x * K + threadIdx.x;
	int j = blockIdx.y * K + threadIdx.y;

	out[j + i * N] = in[i + j * N]; 
}

// One thread per element, in (tilesize)x(tilesize) threadblocks
// thread blocks read & write tiles, in coalesced fashion
// adjacent threads read adjacent input elements, write adjacent output elements
__global__ void transpose_parallel_per_element_tiled(float in[], float out[])
{
	// (i,j) locations of the tile corners for input & output matrices:
	int in_corner_i = blockIdx.x * K, in_corner_j = blockIdx.y * K;
	int out_corner_i = blockIdx.y * K, out_corner_j = blockIdx.x * K;

	int x = threadIdx.x, y = threadIdx.y;

	__shared__ float tile[K][K];

	// Coalesced read from global memory and transposed write into shared memory
	tile[y][x] = in[(in_corner_i + x) + (in_corner_j + y) * N];
	__syncthreads();
	// Read from shared memory and coalesced write to global memory
	out[(out_corner_i + x) + (out_corner_j + y) * N] = tile[x][y];
}

// One thread per element, in (tilesize)x(tilesize) threadblocks
// thread blocks read & write tiles, in coalesced fashion
// adjacent threads read adjacent input elements, write adjacent output elmts
__global__ void transpose_parallel_per_element_tiled16(float in[], float out[])
{
	// (i,j) locations of the tile corners for input & output matrices:
	int in_corner_i = blockIdx.x * 16, in_corner_j = blockIdx.y * 16;
	int out_corner_i = blockIdx.y * 16, out_corner_j = blockIdx.x * 16;

	int x = threadIdx.x, y = threadIdx.y;

	__shared__ float tile[16][16];

	// Coalesced read from global memory and transposed write into shared memory
	tile[y][x] = in[(in_corner_i + x) + (in_corner_j + y) * N];
	__syncthreads();
	// Read from shared memory and coalesced write to global memory
	out[(out_corner_i + x) + (out_corner_j + y) * N] = tile[x][y];
}

// One thread per element, in KxK threadblocks
// thread blocks read & write tiles, in coalesced fashion
// shared memory array has been padded to avoid bank conflicts
__global__ void transpose_parallel_per_element_tiled_padded(float in[], float out[])
{
	// (i,j) locations of the tile corners for input & output matrices:
	int in_corner_i = blockIdx.x * K, in_corner_j = blockIdx.y * K;
	int out_corner_i = blockIdx.y * K, out_corner_j = blockIdx.x * K;

	int x = threadIdx.x, y = threadIdx.y;

	__shared__ float tile[K][K + 1];

	// Coalesced read from global memory and transposed write into shared memory
	tile[y][x] = in[(in_corner_i + x) + (in_corner_j + y) * N];
	__syncthreads();
	// Read from shared memory and coalesced write to global memory
	out[(out_corner_i + x) + (out_corner_j + y) * N] = tile[x][y];
}

// One thread per element, in 16x16 threadblocks
// thread blocks read & write tiles, in coalesced fashion
// shared memory array has been padded to avoid bank conflicts
__global__ void transpose_parallel_per_element_tiled_padded16(float in[], float out[])
{
	// (i,j) locations of the tile corners for input & output matrices:
	int in_corner_i = blockIdx.x * 16, in_corner_j = blockIdx.y * 16;
	int out_corner_i = blockIdx.y * 16, out_corner_j = blockIdx.x * 16;

	int x = threadIdx.x, y = threadIdx.y;

	__shared__ float tile[16][16 + 1];

	// Coalesced read from global memory and transposed write into shared memory
	tile[y][x] = in[(in_corner_i + x) + (in_corner_j + y) * N];
	__syncthreads();
	// Read from shared memory and coalesced write to global memory
	out[(out_corner_i + x) + (out_corner_j + y) * N] = tile[x][y];
}

int main()
{
	int numbytes = N * N * sizeof(float);

	float *in = (float*)malloc(numbytes);
	float *out = (float*)malloc(numbytes);
	float *reference_output = (float*)malloc(numbytes);

	fill_matrix(in); 

	printf("The matrix used is of size (%d, %d)\n", N, N);	

	clock_t start, end;
	double cpu_time_used, gpu_time_used;
     
     	start = clock();
	transpose_CPU(in, reference_output); // This creates the reference matrix (the expected output)
	end = clock();
	cpu_time_used = ((double) (end - start) * 1000) / CLOCKS_PER_SEC;
	printf("transpose_CPU                                                : Finished in %lf ms\n", cpu_time_used);

	float* d_in, * d_out;

	hipMalloc(&d_in, numbytes);
	hipMalloc(&d_out, numbytes);
	hipMemcpy(d_in, in, numbytes, hipMemcpyHostToDevice);

	// Transpose with each thread taking care of one row in the matrix
     	start = clock();
	transpose_parallel_per_row <<<1, N>>> (d_in, d_out);
	end = clock();
	gpu_time_used = ((double) (end - start) * 1000) / CLOCKS_PER_SEC;
	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);	
	printf("transpose_parallel_per_row                                   : Finished in %lf ms and the output obtained is: %s\n", gpu_time_used, compare_matrices(out, reference_output) ? "Incorrect" : "Correct");

	
     	start = clock();
	transpose_serial <<<1, 1>>> (d_in, d_out);
	end = clock();
	gpu_time_used = ((double) (end - start) * 1000) / CLOCKS_PER_SEC;

	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
	
	printf("transpose_serial                                             : Finished in %lf ms and the output obtained is: %s\n", gpu_time_used, compare_matrices(out, reference_output) ? "Incorrect" : "Correct");



	dim3 blocks(N / K, N / K); // blocks per grid
	dim3 threads(K, K);	// threads per block

	// Transpose with a thread for each element in the matrix
     	start = clock();
	transpose_parallel_per_element <<<blocks, threads>>> (d_in, d_out);
	end = clock();
	gpu_time_used = ((double) (end - start) * 1000) / CLOCKS_PER_SEC;
	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
	printf("transpose_parallel_per_element                               : Finished in %lf ms and the output obtained is: %s\n", gpu_time_used, compare_matrices(out, reference_output) ? "Incorrect" : "Correct");


	// Tiled transpose with a thread for each element in the tile
     	start = clock();
	transpose_parallel_per_element_tiled <<<blocks, threads>>> (d_in, d_out);
	end = clock();
	gpu_time_used = ((double) (end - start) * 1000) / CLOCKS_PER_SEC;
	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
	printf("transpose_parallel_per_element_tiled (32X32)                 : Finished in %lf ms and the output obtained is: %s\n", gpu_time_used, compare_matrices(out, reference_output) ? "Incorrect" : "Correct");



	dim3 blocks16x16(N / 16, N / 16); // blocks per grid
	dim3 threads16x16(16, 16);	 // threads per block


	// Tiled transpose with each element of the tile given to a thread
     	start = clock();
	transpose_parallel_per_element_tiled16 <<<blocks16x16, threads16x16>>> (d_in, d_out);
	end = clock();
	gpu_time_used = ((double) (end - start) * 1000) / CLOCKS_PER_SEC;
	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
	printf("transpose_parallel_per_element_tiled (16X16)                 : Finished in %lf ms and the output obtained is: %s\n", gpu_time_used, compare_matrices(out, reference_output) ? "Incorrect" : "Correct");


	// Tiled transpose with padding
     	start = clock();
	transpose_parallel_per_element_tiled_padded16 <<<blocks16x16, threads16x16>>> (d_in, d_out);
	end = clock();
	gpu_time_used = ((double) (end - start) * 1000) / CLOCKS_PER_SEC;
	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);	
	printf("transpose_parallel_per_element_tiled_padded (16X16)          : Finished in %lf ms and the output obtained is: %s\n", gpu_time_used, compare_matrices(out, reference_output) ? "Incorrect" : "Correct");


	hipFree(d_in);
	hipFree(d_out);
}

