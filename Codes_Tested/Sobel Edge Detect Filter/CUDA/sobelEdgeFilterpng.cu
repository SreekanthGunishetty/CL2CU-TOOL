#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include <math.h>
#include "lodepng.h"
#include <string>
#include <time.h>
#define N 20.0

// nvcc sobelEdgeFilterpng.cu lodepng.cpp -arch sm_20
// Time taken by CPU :  0.95400 ms
// Time taken by GPU :  0.09100 ms


typedef unsigned char byte;

// making a structure for the image with pixels, width and height
struct imgData {
  byte* pixels;
  unsigned int width;
  unsigned int height;
};

imgData loadImage(char* filename) {
  unsigned int width, height;
  byte* rgb;
  unsigned error = lodepng_decode_file(&rgb, &width, &height, filename, LCT_RGBA, 8);
  if (error) {
    printf("LodePNG had an error during file processing. Exiting program.\n");
    // printf("Error code: %u: %s\n", error, lodepng_error_text(error));
    exit(2);
  }
  // we convert to grayscale here
  byte* grayscale = new byte[width * height];
  // pixels stored as rgba
  byte* img = rgb;
  for (int i = 0; i < width * height; ++i) {
    int r = *img++;
    int g = *img++;
    int b = *img++;
    int a = *img++;
    grayscale[i] = 0.3 * r + 0.6 * g + 0.1 * b + 0.5;
  }
  free(rgb);
  imgData a;
  a.pixels = grayscale;a.width = width;a.height = height;
  return a;
}


void writeImage(char* filename, imgData img) {
  std::string newName = filename;
  // checking image coordinates
  // printf("starting to write %d is width and %d is height\n", img.width, img.height);
  unsigned error = lodepng_encode_file(newName.c_str(), img.pixels, img.width, img.height, LCT_GREY, 8);
  if (error) {
    printf("LodePNG had an error during file writing. Exiting program.\n");
    exit(3);
  }
  else
  {
    printf("No error while writing?\n");
  }
  delete [] img.pixels;
}



void sobelCpu(byte* orig, byte* op, const unsigned int width, const unsigned int height) {
  for (int y = 1; y < height - 1; y++) {
    for (int x = 1; x < width - 1; x++) {
      // we find dx and dy
      int dx = (-1 * orig[(y - 1) * width + (x - 1)]) + (-2 * orig[y * width + (x - 1)]) + (-1 * orig[(y + 1) * width + (x - 1)]) +
      (orig[(y - 1) * width + (x + 1)]) + (2 * orig[y * width + (x + 1)]) + (orig[(y + 1) * width + (x + 1)]);
      int dy = (orig[(y - 1) * width + (x - 1)]) + (2 * orig[(y - 1) * width + x]) + (orig[(y - 1) * width + (x + 1)]) +
      (-1 * orig[(y + 1) * width + (x - 1)]) + (-2 * orig[(y + 1) * width + x]) + (-1 * orig[(y + 1) * width + (x + 1)]);
      // sqrt(dx^2 + dy^2) gives the new pixel value
      op[y * width + x] = sqrt((float)((dx * dx) + (dy * dy)));
    }
  }
}

__global__ void sobelGpu(const byte* orig, byte* gpu, const unsigned int width, const unsigned int height) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  float dx, dy;
  if( x > 0 && y > 0 && x < (width-1) && y < (height-1)) {
    dx = (-1* orig[(y-1)*width + (x-1)]) + (-2*orig[y*width+(x-1)]) + (-1*orig[(y+1)*width+(x-1)]) +
    (    orig[(y-1)*width + (x+1)]) + ( 2*orig[y*width+(x+1)]) + (   orig[(y+1)*width+(x+1)]);
    dy = (    orig[(y-1)*width + (x-1)]) + ( 2*orig[(y-1)*width+x]) + (   orig[(y-1)*width+(x+1)]) +
    (-1* orig[(y+1)*width + (x-1)]) + (-2*orig[(y+1)*width+x]) + (-1*orig[(y+1)*width+(x+1)]);
    gpu[y*width + x] = sqrt( (dx*dx) + (dy*dy) );
  }
}



int main()
{
  byte *d_in,*d_out;

  imgData in = loadImage("person.png");
  
  int w = in.width;int h=in.height;
  
  // checking dimensions
  // printf("width is %d, height is %d\n",w,h);
  
  // imgData op(new byte[w*h],w,h);
  // imgData op2(new byte[w*h],w,h);
  imgData op;op.pixels = (byte*)malloc(sizeof(byte)*w*h);
  op.width=w;op.height=h;

  imgData op2;op2.pixels = (byte*)malloc(sizeof(byte)*w*h);
  op2.width=w;op.height=h;


  // debuggin to check about the 
  // writeImage("personcameas.png", in);

  hipMalloc((void **)&d_in,(w*h));
  hipMalloc((void **)&d_out,(w*h));

  hipMemcpy(d_in,in.pixels,w*h,hipMemcpyHostToDevice);
  hipMemset(d_out, 0, w*h);

  dim3 threadsPerBlock(N,N,1);
  dim3 numBlocks(ceil(w/N),ceil(h/N),1);


  // struct timeval t1, t2;
  // gettimeofday(&t1, 0);
  sobelCpu(in.pixels,op.pixels,w,h);
  // gettimeofday(&t2, 0);

  // double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;

  // printf("Time taken by CPU :  %3.5f ms \n", time);
  
  // printf("before writing to cpuimg\n");
  writeImage("fromcpu.png", op);

  // printf("after writing to cpuimg\n");
  // writeImage("aftercpu.png", in);
  
  // gettimeofday(&t1, 0);
  sobelGpu<<<numBlocks,threadsPerBlock>>>(d_in,d_out,w,h);
  
  // checking if any errors occured
  hipError_t cudaerror = hipDeviceSynchronize(); // waits for completion, returns error code
  if ( hipDeviceSynchronize() != hipSuccess ) fprintf( stderr, "Cuda failed to synchronize:\n"); // if error, output error
  // gettimeofday(&t2, 0);
  
  // printf("survived kernel\n");
  // time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
  // printf("Time taken by GPU :  %3.5f ms \n", time);

  // writeImage("aftergpucpupersoncameas.png", in);
  hipMemcpy(op2.pixels,d_out,(w*h),hipMemcpyDeviceToHost);
  op2.width=w;
  op2.height=h;
  
  writeImage("fromgpu.png", op2);
  
}

