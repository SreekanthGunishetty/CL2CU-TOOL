#include "hip/hip_runtime.h"
#include <float.h>
#include <math.h>
#include <stdio.h>

#include "utils.h"

#include <iostream>
#include <string>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void naive_normalized_cross_correlation(
    float* d_response,
    unsigned char* d_original,
    unsigned char* d_template,
    int            num_pixels_y,
    int            num_pixels_x,
    int            template_half_height,
    int            template_height,
    int            template_half_width,
    int            template_width,
    int            template_size,
    float          template_mean
)
{
    int  ny = num_pixels_y;
    int  nx = num_pixels_x;
    int  knx = template_width;
    int2 image_index_2d = make_int2((blockIdx.x * blockDim.x) + threadIdx.x, (blockIdx.y * blockDim.y) + threadIdx.y);
    int  image_index_1d = (nx * image_index_2d.y) + image_index_2d.x;

    if (image_index_2d.x < nx && image_index_2d.y < ny)
    {
        float image_sum = 0.0f;

        for (int y = -template_half_height; y <= template_half_height; y++)
        {
            for (int x = -template_half_width; x <= template_half_width; x++)
            {
                int2 image_offset_index_2d = make_int2(image_index_2d.x + x, image_index_2d.y + y);
                int2 image_offset_index_2d_clamped = make_int2(min(nx - 1, max(0, image_offset_index_2d.x)), min(ny - 1, max(0, image_offset_index_2d.y)));
                int  image_offset_index_1d_clamped = (nx * image_offset_index_2d_clamped.y) + image_offset_index_2d_clamped.x;

                unsigned char image_offset_value = d_original[image_offset_index_1d_clamped];

                image_sum += (float)image_offset_value;
            }
        }

        float image_mean = image_sum / (float)template_size;

        float sum_of_image_template_diff_products = 0.0f;
        float sum_of_squared_image_diffs = 0.0f;
        float sum_of_squared_template_diffs = 0.0f;

        for (int y = -template_half_height; y <= template_half_height; y++)
        {
            for (int x = -template_half_width; x <= template_half_width; x++)
            {
                int2 image_offset_index_2d = make_int2(image_index_2d.x + x, image_index_2d.y + y);
                int2 image_offset_index_2d_clamped = make_int2(min(nx - 1, max(0, image_offset_index_2d.x)), min(ny - 1, max(0, image_offset_index_2d.y)));
                int  image_offset_index_1d_clamped = (nx * image_offset_index_2d_clamped.y) + image_offset_index_2d_clamped.x;

                unsigned char image_offset_value = d_original[image_offset_index_1d_clamped];
                float         image_diff = (float)image_offset_value - image_mean;

                int2 template_index_2d = make_int2(x + template_half_width, y + template_half_height);
                int  template_index_1d = (knx * template_index_2d.y) + template_index_2d.x;

                unsigned char template_value = d_template[template_index_1d];
                float         template_diff = template_value - template_mean;

                float image_template_diff_product = image_offset_value * template_diff;
                float squared_image_diff = image_diff * image_diff;
                float squared_template_diff = template_diff * template_diff;

                sum_of_image_template_diff_products += image_template_diff_product;
                sum_of_squared_image_diffs += squared_image_diff;
                sum_of_squared_template_diffs += squared_template_diff;
            }
        }

        float result_value = 0.0f;

        if (sum_of_squared_image_diffs != 0 && sum_of_squared_template_diffs != 0)
        {
            result_value = sum_of_image_template_diff_products / sqrt(sum_of_squared_image_diffs * sum_of_squared_template_diffs);
        }

        d_response[image_index_1d] = result_value;
    }
}


__global__ void remove_redness_from_coordinates(
    const unsigned int* d_coordinates,
    unsigned char* d_r,
    unsigned char* d_b,
    unsigned char* d_g,
    unsigned char* d_r_output,
    int    num_coordinates,
    int    num_pixels_y,
    int    num_pixels_x,
    int    template_half_height,
    int    template_half_width
)
{
    int ny = num_pixels_y;
    int nx = num_pixels_x;
    int global_index_1d = (blockIdx.x * blockDim.x) + threadIdx.x;

    int imgSize = num_pixels_x * num_pixels_y;

    if (global_index_1d < num_coordinates)
    {
        unsigned int image_index_1d = d_coordinates[imgSize - global_index_1d - 1];
        ushort2 image_index_2d = make_ushort2(image_index_1d % num_pixels_x, image_index_1d / num_pixels_x);

        for (int y = image_index_2d.y - template_half_height; y <= image_index_2d.y + template_half_height; y++)
        {
            for (int x = image_index_2d.x - template_half_width; x <= image_index_2d.x + template_half_width; x++)
            {
                int2 image_offset_index_2d = make_int2(x, y);
                int2 image_offset_index_2d_clamped = make_int2(min(nx - 1, max(0, image_offset_index_2d.x)), min(ny - 1, max(0, image_offset_index_2d.y)));
                int  image_offset_index_1d_clamped = (nx * image_offset_index_2d_clamped.y) + image_offset_index_2d_clamped.x;

                unsigned char g_value = d_g[image_offset_index_1d_clamped];
                unsigned char b_value = d_b[image_offset_index_1d_clamped];

                unsigned int gb_average = (g_value + b_value) / 20;
                //printf("heya\t");
                d_r_output[image_offset_index_1d_clamped] = (unsigned char)gb_average;
            }
        }
    }
}




static size_t numRowsImg;
static size_t numColsImg;
static size_t templateHalfWidth;
static size_t templateHalfHeight;

static uchar4* inImg;
static uchar4* eyeTemplate;



void loadImageHDR(const std::string& filename,
    float** imagePtr,
    size_t* numRows, size_t* numCols)
{
    cv::Mat image = cv::imread(filename.c_str(), cv::IMREAD_COLOR | cv::IMREAD_ANYDEPTH);
    if (image.empty()) {
        std::cerr << "Couldn't open file: " << filename << std::endl;
        exit(1);
    }

    if (image.channels() != 3) {
        std::cerr << "Image must be color!" << std::endl;
        exit(1);
    }

    if (!image.isContinuous()) {
        std::cerr << "Image isn't continuous!" << std::endl;
        exit(1);
    }

    *imagePtr = new float[image.rows * image.cols * image.channels()];

    float* cvPtr = image.ptr<float>(0);
    for (size_t i = 0; i < image.rows * image.cols * image.channels(); ++i)
        (*imagePtr)[i] = cvPtr[i];

    *numRows = image.rows;
    *numCols = image.cols;
}

void loadImageRGBA(const std::string& filename,
    uchar4** imagePtr,
    size_t* numRows, size_t* numCols)
{
    cv::Mat image = cv::imread(filename);
    if (image.empty()) {
        std::cerr << "Couldn't open file: " << filename << std::endl;
        exit(1);
    }

    if (image.channels() != 3) {
        std::cerr << "Image must be color!" << std::endl;
        exit(1);
    }

    if (!image.isContinuous()) {
        std::cerr << "Image isn't continuous!" << std::endl;
        exit(1);
    }

    cv::Mat imageRGBA;
    cv::cvtColor(image, imageRGBA, cv::COLOR_BGR2RGBA);

    *imagePtr = new uchar4[image.rows * image.cols];

    unsigned char* cvPtr = imageRGBA.ptr<unsigned char>(0);
    for (size_t i = 0; i < image.rows * image.cols; ++i) {
        (*imagePtr)[i].x = cvPtr[4 * i + 0];
        (*imagePtr)[i].y = cvPtr[4 * i + 1];
        (*imagePtr)[i].z = cvPtr[4 * i + 2];
        (*imagePtr)[i].w = cvPtr[4 * i + 3];
    }

    *numRows = image.rows;
    *numCols = image.cols;
}

void saveImageRGBA(const uchar4* const image,
    const size_t numRows, const size_t numCols,
    const std::string& output_file)
{
    int sizes[2];
    sizes[0] = numRows;
    sizes[1] = numCols;
    cv::Mat imageRGBA(2, sizes, CV_8UC4, (void*)image);
    cv::Mat imageOutputBGR;
    cv::cvtColor(imageRGBA, imageOutputBGR, cv::COLOR_RGBA2BGR);
    //output the image
    cv::imwrite(output_file.c_str(), imageOutputBGR);
}

//output an exr file
//assumed to already be BGR
void saveImageHDR(const float* const image,
    const size_t numRows, const size_t numCols,
    const std::string& output_file)
{
    int sizes[2];
    sizes[0] = numRows;
    sizes[1] = numCols;

    cv::Mat imageHDR(2, sizes, CV_32FC3, (void*)image);

    imageHDR = imageHDR * 255;

    cv::imwrite(output_file.c_str(), imageHDR);
}

void CPU_radix(unsigned int* inputVals,
    unsigned int* inputPos,
    unsigned int* outputVals,
    unsigned int* outputPos,
    const size_t numElems)
{
    const int numBits = 1;
    const int numBins = 1 << numBits;

    unsigned int* binHistogram = new unsigned int[numBins];
    unsigned int* binScan = new unsigned int[numBins];

    unsigned int* vals_src = inputVals;
    unsigned int* pos_src = inputPos;

    unsigned int* vals_dst = outputVals;
    unsigned int* pos_dst = outputPos;

    for (unsigned int i = 0; i < 8 * sizeof(unsigned int); i += numBits) {
        unsigned int mask = (numBins - 1) << i;

        memset(binHistogram, 0, sizeof(unsigned int) * numBins);
        memset(binScan, 0, sizeof(unsigned int) * numBins);

        //perform histogram of data & mask into bins
        for (unsigned int j = 0; j < numElems; ++j) {
            // printf("%d ",vals_src[j]);
            unsigned int bin = (vals_src[j] & mask) >> i;
            binHistogram[bin]++;
        }

        //perform exclusive prefix sum (scan) on binHistogram to get starting
        //location for each bin
        for (unsigned int j = 1; j < numBins; ++j) {
            binScan[j] = binScan[j - 1] + binHistogram[j - 1];
        }

        //Gather everything into the correct location
        //need to move vals and positions
        for (unsigned int j = 0; j < numElems; ++j) {
            unsigned int bin = (vals_src[j] & mask) >> i;
            vals_dst[binScan[bin]] = vals_src[j];
            pos_dst[binScan[bin]] = pos_src[j];
            binScan[bin]++;
        }

        std::swap(vals_dst, vals_src);
        std::swap(pos_dst, pos_src);
    }

    std::copy(inputVals, inputVals + numElems, outputVals);
    std::copy(inputPos, inputPos + numElems, outputPos);

    delete[] binHistogram;
    delete[] binScan;
}

int main() {
    unsigned int* inputVals;
    unsigned int* inputPos;
    unsigned int* outputVals;
    unsigned int* outputPos;

    size_t numElems;


    std::string input_file = "red_eye_effect_5.jpg";
    std::string template_file = "red_eye_effect_template_5.jpg";
    std::string output_file = "fromGPU.jpg";
    std::string reference_file = "fromCPU.jpg";


    // thrust::device_vector<unsigned char> d_red;
    // thrust::device_vector<unsigned char> d_blue;
    // thrust::device_vector<unsigned char> d_green;



    size_t numRowsTemplate, numColsTemplate, numRowsImg, nowColsImg;

    loadImageRGBA(input_file, &inImg, &numRowsImg, &numColsImg);
    loadImageRGBA(template_file, &eyeTemplate, &numRowsTemplate, &numColsTemplate);

    templateHalfWidth = (numColsTemplate - 1) / 2;
    templateHalfHeight = (numRowsTemplate - 1) / 2;

    //we need to split each image into its separate channels
    //use thrust to demonstrate basic uses

    numElems = numRowsImg * numColsImg;
    size_t templateSize = numRowsTemplate * numColsTemplate;

    uchar* r = new uchar[numElems];
    uchar* g = new uchar[numElems];
    uchar* b = new uchar[numElems];

    uchar* d_r, * d_b, * d_g;

    uchar* d_op_r;

    hipMalloc((void**)&d_r, sizeof(uchar) * numElems);
    hipMalloc((void**)&d_op_r, sizeof(uchar) * numElems);
    hipMalloc((void**)&d_g, sizeof(uchar) * numElems);
    hipMalloc((void**)&d_b, sizeof(uchar) * numElems);
    for (size_t i = 0; i < numRowsImg * numColsImg; ++i)
    {
        r[i] = (inImg[i].x);
        g[i] = (inImg[i].y);
        b[i] = (inImg[i].z);
    }
    hipMemcpy(d_r, r, sizeof(uchar) * numElems, hipMemcpyHostToDevice);
    hipMemcpy(d_op_r, b, sizeof(uchar) * numElems, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(uchar) * numElems, hipMemcpyHostToDevice);
    hipMemcpy(d_g, g, sizeof(uchar) * numElems, hipMemcpyHostToDevice);
    uchar* rt = new uchar[templateSize];
    uchar* gt = new uchar[templateSize];
    uchar* bt = new uchar[templateSize];
    //hipMalloc((void**)&r,numElems);
    //hipMalloc((void**)&g, numElems);
    //hipMalloc((void**)&b, numElems);
    for (size_t i = 0; i < templateSize; ++i)
    {
        rt[i] = (eyeTemplate[i].x);
        gt[i] = (eyeTemplate[i].y);
        bt[i] = (eyeTemplate[i].z);
    }
    uchar* d_rt, * d_bt, * d_gt;

    hipMalloc((void**)&d_rt, sizeof(uchar) * templateSize);
    hipMalloc((void**)&d_gt, sizeof(uchar) * templateSize);
    hipMalloc((void**)&d_bt, sizeof(uchar) * templateSize);
    hipMemcpy(d_rt, r, sizeof(uchar) * templateSize, hipMemcpyHostToDevice);
    hipMemcpy(d_bt, b, sizeof(uchar) * templateSize, hipMemcpyHostToDevice);
    hipMemcpy(d_gt, g, sizeof(uchar) * templateSize, hipMemcpyHostToDevice);

    unsigned int r_sum, b_sum, g_sum;
    r_sum = 0;b_sum = 0;g_sum = 0;
    for (int i = 0;i < numElems;i++)
    {
        r_sum += r[i];
        b_sum += b[i];
        g_sum += g[i];
    }
    unsigned int rt_sum, bt_sum, gt_sum;
    rt_sum = 0;bt_sum = 0;gt_sum = 0;
    for (int i = 0;i < templateSize;i++)
    {
        rt_sum += rt[i];
        bt_sum += bt[i];
        gt_sum += gt[i];
    }

    float r_mean = (double)rt_sum / templateSize;
    float b_mean = (double)bt_sum / templateSize;
    float g_mean = (double)gt_sum / templateSize;

   // printf("this is rmean\n", r_mean);
    //printf("It came through\n");


    const dim3 blockSize(32, 8, 1);
    const dim3 gridSize((numColsImg + blockSize.x - 1) / blockSize.x, (numRowsImg + blockSize.y - 1) / blockSize.y, 1);

    //now compute the cross-correlations for each channel
    float* red_data;
    hipMalloc((void**)&red_data, sizeof(float) * numElems);
    float* blue_data;
    hipMalloc((void**)&blue_data, sizeof(float) * numElems);
    float* green_data;
    hipMalloc((void**)&green_data, sizeof(float) * numElems);

    naive_normalized_cross_correlation << <gridSize, blockSize >> > (red_data,
        d_r,
        d_rt,
        numRowsImg, numColsImg,
        templateHalfHeight, numRowsTemplate,
        templateHalfWidth, numColsTemplate,
        numRowsTemplate * numColsTemplate, r_mean);

    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());
    // printf("I am still okay\n");

    naive_normalized_cross_correlation << <gridSize, blockSize >> > (blue_data,
        d_b,
        d_bt,
        numRowsImg, numColsImg,
        templateHalfHeight, numRowsTemplate,
        templateHalfWidth, numColsTemplate,
        numRowsTemplate * numColsTemplate, b_mean);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    naive_normalized_cross_correlation << <gridSize, blockSize >> > (green_data,
        d_g,
        d_gt,
        numRowsImg, numColsImg,
        templateHalfHeight, numRowsTemplate,
        templateHalfWidth, numColsTemplate,
        numRowsTemplate * numColsTemplate, g_mean);

    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    float* h_red_data, * h_blue_data, * h_green_data;
    h_red_data = new float[numElems];
    h_green_data = new float[numElems];
    h_blue_data = new float[numElems];
    checkCudaErrors(hipMemcpy(h_red_data, red_data, sizeof(float) * numElems, hipMemcpyDeviceToHost));
    hipMemcpy(h_blue_data, blue_data, sizeof(float) * numElems, hipMemcpyDeviceToHost);
    hipMemcpy(h_green_data, green_data, sizeof(float) * numElems, hipMemcpyDeviceToHost);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    float* combined = new float[numElems];
    float mini = 0;




    for (int i = 0;i < numElems;i++)
    {
        //printf("%f is hred_data",h_red_data[i]);
        combined[i] = h_red_data[i] * h_blue_data[i] * h_green_data[i];
        if (mini > combined[i])
        {
            mini = combined[i];
            //printf("%f is mini", mini);
        }
        // printf("sam\t");
    }
    printf("%f is mini", mini);
    // find min and add bias


    inputVals = new unsigned int[numElems];
    for (int i = 0;i < numElems;i++)
    {
        //printf("combined val: %d \t", combined[i]);
        combined[i] = (double)combined[i] + (double)(-1 * mini);
        //printf("combined val: %f \t", combined[i]);
        inputVals[i] = combined[i];
    }


    inputPos = new unsigned int[numElems];
    //inputVals = (unsigned int*)thrust::raw_pointer_cast(d_combined_response.data());

    for (int i = 0;i < numElems;i++)
    {
        inputPos[i] = i;
    }

    outputVals = new unsigned int[numElems];
    outputPos = new unsigned int[numElems];
    // printf("before radix");
    CPU_radix(inputVals, inputPos, outputVals, outputPos, numElems);
    // printf("after radix");

    const dim3 block2Size(256, 1, 1);
    const dim3 grid2Size((40 + blockSize.x - 1) / blockSize.x, 1, 1);

    unsigned int* d_outputPos;
    hipMalloc((void**)&d_outputPos, sizeof(unsigned int) * numElems);
    hipMemcpy(d_outputPos, outputPos, sizeof(unsigned int) * numElems, hipMemcpyHostToDevice);

    remove_redness_from_coordinates << <grid2Size, block2Size >> > (d_outputPos,
        d_r,
        d_b,
        d_g,
        d_op_r,
        40,
        numRowsImg, numColsImg,
        9, 9);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());


    uchar* h_op_r = new uchar[numElems];
    hipMemcpy(h_op_r, d_op_r, sizeof(uchar) * numElems, hipMemcpyDeviceToHost);
    printf("after the kernel\n");

    // combine channels
    uchar4* outputImg = new uchar4[numElems];
    for (int i = 0;i < numElems;i++)
    {
        outputImg[i].x = h_op_r[i];
        outputImg[i].y = g[i];
        outputImg[i].z = b[i];
        outputImg[i].w = 255;
    }


    saveImageRGBA(outputImg, numRowsImg, numColsImg, reference_file);

    return 0;
}
