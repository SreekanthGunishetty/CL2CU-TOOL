#include "hip/hip_runtime.h"
#include <float.h>
#include <math.h>
#include <stdio.h>

// #include "utils.h"

#include <iostream>
#include <string>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// #include "loadSaveImage.h"
#include <stdio.h>

__global__ void naive_normalized_cross_correlation(
    float* d_response,
    unsigned char* d_original,
    unsigned char* d_template,
    int            num_pixels_y,
    int            num_pixels_x,
    int            template_half_height,
    int            template_height,
    int            template_half_width,
    int            template_width,
    int            template_size,
    float          template_mean
)
{
    int  ny = num_pixels_y;
    int  nx = num_pixels_x;
    int  knx = template_width;
    int2 image_index_2d = make_int2((blockIdx.x * blockDim.x) + threadIdx.x, (blockIdx.y * blockDim.y) + threadIdx.y);
    int  image_index_1d = (nx * image_index_2d.y) + image_index_2d.x;

    if (image_index_2d.x < nx && image_index_2d.y < ny)
    {
        //
        // compute image mean
        //
        float image_sum = 0.0f;

        for (int y = -template_half_height; y <= template_half_height; y++)
        {
            for (int x = -template_half_width; x <= template_half_width; x++)
            {
                int2 image_offset_index_2d = make_int2(image_index_2d.x + x, image_index_2d.y + y);
                int2 image_offset_index_2d_clamped = make_int2(min(nx - 1, max(0, image_offset_index_2d.x)), min(ny - 1, max(0, image_offset_index_2d.y)));
                int  image_offset_index_1d_clamped = (nx * image_offset_index_2d_clamped.y) + image_offset_index_2d_clamped.x;

                unsigned char image_offset_value = d_original[image_offset_index_1d_clamped];

                image_sum += (float)image_offset_value;
            }
        }

        float image_mean = image_sum / (float)template_size;

        //
        // compute sums
        //
        float sum_of_image_template_diff_products = 0.0f;
        float sum_of_squared_image_diffs = 0.0f;
        float sum_of_squared_template_diffs = 0.0f;

        for (int y = -template_half_height; y <= template_half_height; y++)
        {
            for (int x = -template_half_width; x <= template_half_width; x++)
            {
                int2 image_offset_index_2d = make_int2(image_index_2d.x + x, image_index_2d.y + y);
                int2 image_offset_index_2d_clamped = make_int2(min(nx - 1, max(0, image_offset_index_2d.x)), min(ny - 1, max(0, image_offset_index_2d.y)));
                int  image_offset_index_1d_clamped = (nx * image_offset_index_2d_clamped.y) + image_offset_index_2d_clamped.x;

                unsigned char image_offset_value = d_original[image_offset_index_1d_clamped];
                float         image_diff = (float)image_offset_value - image_mean;

                int2 template_index_2d = make_int2(x + template_half_width, y + template_half_height);
                int  template_index_1d = (knx * template_index_2d.y) + template_index_2d.x;

                unsigned char template_value = d_template[template_index_1d];
                float         template_diff = template_value - template_mean;

                float image_template_diff_product = image_offset_value * template_diff;
                float squared_image_diff = image_diff * image_diff;
                float squared_template_diff = template_diff * template_diff;

                sum_of_image_template_diff_products += image_template_diff_product;
                sum_of_squared_image_diffs += squared_image_diff;
                sum_of_squared_template_diffs += squared_template_diff;
            }
        }


        //
        // compute final result
        //
        float result_value = 0.0f;

        if (sum_of_squared_image_diffs != 0 && sum_of_squared_template_diffs != 0)
        {
            result_value = sum_of_image_template_diff_products / sqrt(sum_of_squared_image_diffs * sum_of_squared_template_diffs);
        }

        d_response[image_index_1d] = result_value;
    }
}


__global__ void remove_redness_from_coordinates(
    const unsigned int* d_coordinates,
    unsigned char* d_r,
    unsigned char* d_b,
    unsigned char* d_g,
    unsigned char* d_r_output,
    int    num_coordinates,
    int    num_pixels_y,
    int    num_pixels_x,
    int    template_half_height,
    int    template_half_width
)
{
    int ny = num_pixels_y;
    int nx = num_pixels_x;
    int global_index_1d = (blockIdx.x * blockDim.x) + threadIdx.x;

    int imgSize = num_pixels_x * num_pixels_y;

    if (global_index_1d < num_coordinates)
    {
        unsigned int image_index_1d = d_coordinates[imgSize - global_index_1d - 1];
        ushort2 image_index_2d = make_ushort2(image_index_1d % num_pixels_x, image_index_1d / num_pixels_x);

        for (int y = image_index_2d.y - template_half_height; y <= image_index_2d.y + template_half_height; y++)
        {
            for (int x = image_index_2d.x - template_half_width; x <= image_index_2d.x + template_half_width; x++)
            {
                int2 image_offset_index_2d = make_int2(x, y);
                int2 image_offset_index_2d_clamped = make_int2(min(nx - 1, max(0, image_offset_index_2d.x)), min(ny - 1, max(0, image_offset_index_2d.y)));
                int  image_offset_index_1d_clamped = (nx * image_offset_index_2d_clamped.y) + image_offset_index_2d_clamped.x;

                unsigned char g_value = d_g[image_offset_index_1d_clamped];
                unsigned char b_value = d_b[image_offset_index_1d_clamped];

                unsigned int gb_average = (g_value + b_value) / 2;
                //printf("heya\t");
                d_r_output[image_offset_index_1d_clamped] = (unsigned char)gb_average;
            }
        }
    }
}





static size_t numRowsImg;
static size_t numColsImg;
static size_t templateHalfWidth;
static size_t templateHalfHeight;

static uchar4* inImg;
static uchar4* eyeTemplate;



void loadImageHDR(const std::string& filename,
    float** imagePtr,
    size_t* numRows, size_t* numCols)
{
    cv::Mat image = cv::imread(filename.c_str(), cv::IMREAD_COLOR | cv::IMREAD_ANYDEPTH);
    if (image.empty()) {
        std::cerr << "Couldn't open file: " << filename << std::endl;
        exit(1);
    }

    if (image.channels() != 3) {
        std::cerr << "Image must be color!" << std::endl;
        exit(1);
    }

    if (!image.isContinuous()) {
        std::cerr << "Image isn't continuous!" << std::endl;
        exit(1);
    }

    *imagePtr = new float[image.rows * image.cols * image.channels()];

    float* cvPtr = image.ptr<float>(0);
    for (size_t i = 0; i < image.rows * image.cols * image.channels(); ++i)
        (*imagePtr)[i] = cvPtr[i];

    *numRows = image.rows;
    *numCols = image.cols;
}

void loadImageRGBA(const std::string& filename,
    uchar4** imagePtr,
    size_t* numRows, size_t* numCols)
{
    cv::Mat image = cv::imread(filename);
    if (image.empty()) {
        std::cerr << "Couldn't open file: " << filename << std::endl;
        exit(1);
    }

    if (image.channels() != 3) {
        std::cerr << "Image must be color!" << std::endl;
        exit(1);
    }

    if (!image.isContinuous()) {
        std::cerr << "Image isn't continuous!" << std::endl;
        exit(1);
    }

    cv::Mat imageRGBA;
    cv::cvtColor(image, imageRGBA, cv::COLOR_BGR2RGBA);

    *imagePtr = new uchar4[image.rows * image.cols];

    unsigned char* cvPtr = imageRGBA.ptr<unsigned char>(0);
    for (size_t i = 0; i < image.rows * image.cols; ++i) {
        (*imagePtr)[i].x = cvPtr[4 * i + 0];
        (*imagePtr)[i].y = cvPtr[4 * i + 1];
        (*imagePtr)[i].z = cvPtr[4 * i + 2];
        (*imagePtr)[i].w = cvPtr[4 * i + 3];
    }

    *numRows = image.rows;
    *numCols = image.cols;
}

void saveImageRGBA(const uchar4* const image,
    const size_t numRows, const size_t numCols,
    const std::string& output_file)
{
    int sizes[2];
    sizes[0] = numRows;
    sizes[1] = numCols;
    cv::Mat imageRGBA(2, sizes, CV_8UC4, (void*)image);
    cv::Mat imageOutputBGR;
    cv::cvtColor(imageRGBA, imageOutputBGR, cv::COLOR_RGBA2BGR);
    //output the image
    cv::imwrite(output_file.c_str(), imageOutputBGR);
}

//output an exr file
//assumed to already be BGR
void saveImageHDR(const float* const image,
    const size_t numRows, const size_t numCols,
    const std::string& output_file)
{
    int sizes[2];
    sizes[0] = numRows;
    sizes[1] = numCols;

    cv::Mat imageHDR(2, sizes, CV_32FC3, (void*)image);

    imageHDR = imageHDR * 255;

    cv::imwrite(output_file.c_str(), imageHDR);
}


__global__
void histogram_kernel(unsigned int pass,
                      unsigned int * d_bins,
                      unsigned int* const d_input,
                      const int size) {
  int mid = threadIdx.x + blockDim.x * blockIdx.x;
  if (mid >= size)
    return;
  unsigned int one = 1;
  int bin = ((d_input[mid] & (one << pass)) == (one << pass)) ? 1 : 0;
  if (bin)
    atomicAdd(&d_bins[1], 1);
  else
    atomicAdd(&d_bins[0], 1);
}

// we will run 1 exclusive scan, but then when we
// do the move, for zero vals, we iwll take mid - val of scan there
__global__
void exclusive_scan_kernel(unsigned int pass,
                           unsigned int const * d_inputVals,
                           unsigned int * d_output,
                           const int size,
                           unsigned int base,
                           unsigned int threadSize) {
  int mid = threadIdx.x + threadSize * base;
  int block = threadSize * base;
  unsigned int one = 1;

  if (mid >= size)
    return;

  unsigned int val = 0;
  if (mid > 0)
    val = ((d_inputVals[mid - 1] & (one << pass))  == (one << pass)) ? 1 : 0;
  else
    val = 0;

  d_output[mid] = val;

  __syncthreads();

  for (int s = 1; s <= threadSize; s *= 2) {
    int spot = mid - s;

    if (spot >= 0 && spot >=  threadSize * base)
      val = d_output[spot];
    __syncthreads();
    if (spot >= 0 && spot >= threadSize * base)
      d_output[mid] += val;
    __syncthreads();
  }
  if (base > 0)
    d_output[mid] += d_output[base * threadSize - 1];
}

__global__
void move_kernel(
  unsigned int pass,
  unsigned int* const d_inputVals,
  unsigned int* const d_inputPos,
  unsigned int* d_outputVals,
  unsigned int* d_outputPos,
  unsigned int* d_outputMove,
  unsigned int* const d_scanned,
  unsigned int  one_pos,
  const size_t numElems) {

  int mid = threadIdx.x + blockDim.x * blockIdx.x;
  if (mid >= numElems)
    return;

  unsigned int scan = 0;
  unsigned int base = 0;
  unsigned int one = 1;
  if ( ( d_inputVals[mid] & (one << pass)) == (1 << pass)) {
    scan = d_scanned[mid];
    base = one_pos;
  } else {
    scan = (mid) - d_scanned[mid];
    base = 0;
  }

  d_outputMove[mid] = base + scan;
  d_outputPos[base + scan]  = d_inputPos[mid]; //d_inputPos[0];
  d_outputVals[base + scan] = d_inputVals[mid]; //base+scan;//d_inputVals[0];

}

// max size for n/d better one
int get_max_size(int n, int d) {
  return (int)ceil( (float)n / (float)d ) + 1;
}


// host function for radix sort
void radix_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t numElems)
{
  unsigned int* d_bins;
  unsigned int  h_bins[2];
  unsigned int* d_scanned;
  unsigned int* d_moved;
  const size_t histo_size = 2 * sizeof(unsigned int);
  const size_t arr_size   = numElems * sizeof(unsigned int);

  hipMalloc(&d_bins, histo_size);
  hipMalloc(&d_scanned, arr_size);
  hipMalloc(&d_moved, arr_size);


  // for histogram kernel defined here
  dim3 thread_dim(1024);
  dim3 hist_block_dim(get_max_size(numElems, thread_dim.x));


  for (unsigned int pass = 0; pass < 32; pass++) {
    unsigned int one = 1;
    hipMemset(d_bins, 0, histo_size);
    hipMemset(d_scanned, 0, arr_size);
    hipMemset(d_outputVals, 0, arr_size);
    hipMemset(d_outputPos, 0, arr_size);

    histogram_kernel <<< hist_block_dim, thread_dim>>>(pass, d_bins, d_inputVals, numElems);
    hipDeviceSynchronize();
    // checkCudaErrors(hipGetLastError());

    hipMemcpy(&h_bins, d_bins, histo_size, hipMemcpyDeviceToHost);

    // printf("debugging %d %d %d %d %d \n", h_bins[0], h_bins[1], h_bins[0] + h_bins[1], numElems, (one << pass));

    for (int i = 0; i < get_max_size(numElems, thread_dim.x); i++) {
      exclusive_scan_kernel <<< dim3(1), thread_dim>>>(pass,d_inputVals,d_scanned,numElems,i,thread_dim.x);
      hipDeviceSynchronize();
      // checkCudaErrors(hipGetLastError());
    }
    // calculate the move positions
    move_kernel <<< hist_block_dim, thread_dim>>>(
      pass,
      d_inputVals,
      d_inputPos,
      d_outputVals,
      d_outputPos,
      d_moved,
      d_scanned,
      h_bins[0],
      numElems
    );
    hipDeviceSynchronize(); 
    // checkCudaErrors(hipGetLastError());
    hipMemcpy(d_inputVals, d_outputVals, arr_size, hipMemcpyDeviceToDevice);
    hipMemcpy(d_inputPos, d_outputPos, arr_size, hipMemcpyDeviceToDevice);
    hipDeviceSynchronize(); 
    // checkCudaErrors(hipGetLastError());
  }
  hipFree(d_moved);
  hipFree(d_scanned);
  hipFree(d_bins);
}

int main() {
    unsigned int* inputVals;
    unsigned int* inputPos;
    unsigned int* outputVals;
    unsigned int* outputPos;

    size_t numElems;


    std::string input_file = "red_eye_effect_5.jpg";
    std::string template_file = "red_eye_effect_template_5.jpg";
    std::string output_file = "fromGPU.jpg";
    std::string reference_file = "fromCPU.jpg";

    // thrust::device_vector<unsigned char> d_red;
    // thrust::device_vector<unsigned char> d_blue;
    // thrust::device_vector<unsigned char> d_green;


    size_t numRowsTemplate, numColsTemplate, numRowsImg, nowColsImg;

    loadImageRGBA(input_file, &inImg, &numRowsImg, &numColsImg);
    loadImageRGBA(template_file, &eyeTemplate, &numRowsTemplate, &numColsTemplate);

    templateHalfWidth = (numColsTemplate - 1) / 2;
    templateHalfHeight = (numRowsTemplate - 1) / 2;

    //we need to split each image into its separate channels
    numElems = numRowsImg * numColsImg;
    size_t templateSize = numRowsTemplate * numColsTemplate;



    uchar* r = new uchar[numElems];
    uchar* g = new uchar[numElems];
    uchar* b = new uchar[numElems];

    uchar* d_r, * d_b, * d_g;

    uchar* d_op_r;

    hipMalloc((void**)&d_r, sizeof(uchar) * numElems);
    hipMalloc((void**)&d_op_r, sizeof(uchar) * numElems);
    hipMalloc((void**)&d_g, sizeof(uchar) * numElems);
    hipMalloc((void**)&d_b, sizeof(uchar) * numElems);
    for (size_t i = 0; i < numRowsImg * numColsImg; ++i)
    {
        r[i] = (inImg[i].x);
        g[i] = (inImg[i].y);
        b[i] = (inImg[i].z);
    }
    hipMemcpy(d_r, r, sizeof(uchar) * numElems, hipMemcpyHostToDevice);
    hipMemcpy(d_op_r, b, sizeof(uchar) * numElems, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(uchar) * numElems, hipMemcpyHostToDevice);
    hipMemcpy(d_g, g, sizeof(uchar) * numElems, hipMemcpyHostToDevice);
    uchar* rt = new uchar[templateSize];
    uchar* gt = new uchar[templateSize];
    uchar* bt = new uchar[templateSize];
    //hipMalloc((void**)&r,numElems);
    //hipMalloc((void**)&g, numElems);
    //hipMalloc((void**)&b, numElems);
    for (size_t i = 0; i < templateSize; ++i)
    {
        rt[i] = (eyeTemplate[i].x);
        gt[i] = (eyeTemplate[i].y);
        bt[i] = (eyeTemplate[i].z);
    }
    uchar* d_rt, * d_bt, * d_gt;

    hipMalloc((void**)&d_rt, sizeof(uchar) * templateSize);
    hipMalloc((void**)&d_gt, sizeof(uchar) * templateSize);
    hipMalloc((void**)&d_bt, sizeof(uchar) * templateSize);
    hipMemcpy(d_rt, r, sizeof(uchar) * templateSize, hipMemcpyHostToDevice);
    hipMemcpy(d_bt, b, sizeof(uchar) * templateSize, hipMemcpyHostToDevice);
    hipMemcpy(d_gt, g, sizeof(uchar) * templateSize, hipMemcpyHostToDevice);

    unsigned int r_sum, b_sum, g_sum;
    r_sum = 0;b_sum = 0;g_sum = 0;
    for (int i = 0;i < numElems;i++)
    {
        r_sum += r[i];
        b_sum += b[i];
        g_sum += g[i];
    }
    unsigned int rt_sum, bt_sum, gt_sum;
    rt_sum = 0;bt_sum = 0;gt_sum = 0;
    for (int i = 0;i < templateSize;i++)
    {
        rt_sum += rt[i];
        bt_sum += bt[i];
        gt_sum += gt[i];
    }

    float r_mean = (double)rt_sum / templateSize;
    float b_mean = (double)bt_sum / templateSize;
    float g_mean = (double)gt_sum / templateSize;

    //printf("this is rmean\n", r_mean);
    //printf("It came through\n");


    const dim3 blockSize(32, 8, 1);
    const dim3 gridSize((numColsImg + blockSize.x - 1) / blockSize.x, (numRowsImg + blockSize.y - 1) / blockSize.y, 1);

    //now compute the cross-correlations for each channel
    float* red_data;
    hipMalloc((void**)&red_data, sizeof(float) * numElems);
    float* blue_data;
    hipMalloc((void**)&blue_data, sizeof(float) * numElems);
    float* green_data;
    hipMalloc((void**)&green_data, sizeof(float) * numElems);

    naive_normalized_cross_correlation <<<gridSize, blockSize >> > (red_data,
        d_r,
        d_rt,
        numRowsImg, numColsImg,
        templateHalfHeight, numRowsTemplate,
        templateHalfWidth, numColsTemplate,
        numRowsTemplate * numColsTemplate, r_mean);

    hipDeviceSynchronize();
    // checkCudaErrors(hipGetLastError());

    naive_normalized_cross_correlation << <gridSize, blockSize >> > (blue_data,
        d_b,
        d_bt,
        numRowsImg, numColsImg,
        templateHalfHeight, numRowsTemplate,
        templateHalfWidth, numColsTemplate,
        numRowsTemplate * numColsTemplate, b_mean);
    hipDeviceSynchronize(); 
    // checkCudaErrors(hipGetLastError());

    naive_normalized_cross_correlation << <gridSize, blockSize >> > (green_data,
        d_g,
        d_gt,
        numRowsImg, numColsImg,
        templateHalfHeight, numRowsTemplate,
        templateHalfWidth, numColsTemplate,
        numRowsTemplate * numColsTemplate, g_mean);

    hipDeviceSynchronize();
     // checkCudaErrors(hipGetLastError());

    float* h_red_data, * h_blue_data, * h_green_data;
    h_red_data = new float[numElems];
    h_green_data = new float[numElems];
    h_blue_data = new float[numElems];
    hipMemcpy(h_red_data, red_data, sizeof(float) * numElems, hipMemcpyDeviceToHost);
    hipMemcpy(h_blue_data, blue_data, sizeof(float) * numElems, hipMemcpyDeviceToHost);
    hipMemcpy(h_green_data, green_data, sizeof(float) * numElems, hipMemcpyDeviceToHost);
    hipDeviceSynchronize(); 
    // checkCudaErrors(hipGetLastError());
    float* combined = new float[numElems];
    float mini = 0;




    for (int i = 0;i < numElems;i++)
    {
        combined[i] = h_red_data[i] * h_blue_data[i] * h_green_data[i];
        if (mini > combined[i])
        {
            mini = combined[i];
            //printf("%f is mini", mini);
        }
    }
    printf("%f is mini", mini);
    // find min and add bias so NO VALUE is negetive, easier to deal with


    inputVals = new unsigned int[numElems];
    for (int i = 0;i < numElems;i++)
    {
        //printf("combined val: %d \t", combined[i]);
        combined[i] = (double)combined[i] + (double)(-1 * mini);
        //printf("combined val: %f \t", combined[i]);
        inputVals[i] = combined[i];
    }


    inputPos = new unsigned int[numElems];

    for (int i = 0;i < numElems;i++)
    {
        inputPos[i] = i;
    }

    unsigned int* d_inputVals;
    hipMalloc((void**)&d_inputVals, sizeof(unsigned int)* numElems);
    hipMemcpy(d_inputVals, inputVals, sizeof(unsigned int)* numElems, hipMemcpyHostToDevice);

    unsigned int* d_inputPos;
    hipMalloc((void **)&d_inputPos,sizeof(unsigned int)*numElems);
    hipMemcpy(d_inputPos,inputPos, sizeof(unsigned int) * numElems,hipMemcpyHostToDevice);

    outputVals = new unsigned int[numElems];
    outputPos = new unsigned int[numElems];

    unsigned int* d_outputPos, * d_outputVals;
    hipMalloc((void**)&d_outputPos, sizeof(unsigned int) * numElems);
    hipMalloc((void**)&d_outputVals, sizeof(unsigned int) * numElems);
    // printf("before radix");

    radix_sort(d_inputVals,d_inputPos,d_outputVals,d_outputPos,numElems);

    // printf("after radix");

    const dim3 block2Size(256, 1, 1);
    const dim3 grid2Size((40 + blockSize.x - 1) / blockSize.x, 1, 1);
    /*for (int i = 0;i < 20;i++)
    {
        printf("i:%d and outputPos: %d\n",i,outputPos[i]);
    }*/

    remove_redness_from_coordinates <<<grid2Size, block2Size >>> (d_outputPos,
        d_r,
        d_b,
        d_g,
        d_op_r,
        40,
        numRowsImg, numColsImg,
        9, 9);
    hipDeviceSynchronize(); 
    // checkCudaErrors(hipGetLastError());

    uchar* h_op_r = new uchar[numElems];
    hipMemcpy(h_op_r, d_op_r, sizeof(uchar) * numElems, hipMemcpyDeviceToHost);
    printf("after the kernel\n");

    // combine channels
    uchar4* outputImg = new uchar4[numElems];
    for (int i = 0;i < numElems;i++)
    {
        outputImg[i].x = h_op_r[i];
        outputImg[i].y = g[i];
        outputImg[i].z = b[i];
        outputImg[i].w = 255;
    }


    saveImageRGBA(outputImg, numRowsImg, numColsImg, output_file);

    return 0;
}
