
#include <hip/hip_runtime.h>
#include <stdio.h>
# define N 512


void fill_array(int *data)
{
	for(int i=0;i<N;i++)
		data[i]=i;
}

void host_add(int *a,int* b,int* c)
{
	for(int i=0;i<N;i++)
		c[i] = a[i] + b[i];
}

void print_output(int* op)
{
	for(int i=0;i<N;i++)
		printf("%d\n",op[i]);
}

__global__ void device_add(int* a,int* b,int* c)
{
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	c[index] = a[index] + b[index];
}

int main(void)
{
	int *a,*b,*c;
	int size = N * sizeof(int);
	a= (int*)malloc(size);
	b= (int*)malloc(size);
	c= (int*)malloc(size);
	fill_array(a);
	fill_array(b);

	int *d_a,*d_b,*d_c;
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);


	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);


	device_add <<<2,N/2>>> (d_a,d_b,d_c);
	
	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	// host_add(a,b,c);

	print_output(c);

	free(a);free(b);free(c);
	hipFree(d_a);hipFree(d_b);hipFree(d_c);
return 0;
}
