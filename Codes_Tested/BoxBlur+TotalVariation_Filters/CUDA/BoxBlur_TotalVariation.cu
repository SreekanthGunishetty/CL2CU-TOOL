#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include<time.h>

#include <stdio.h>

//OpenCV stuff
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>
using namespace cv;

// I learnt about vector notaion in c++ and now know that uchar4* is a 2D array of 4 element vectors of unsigned char type.  

__global__ void box_blur(const unsigned char* inputChannel, unsigned char* outputChannel, int rows, int cols, int filterWidth, int factor) // This is for square kernels only
{
	// Calculating the coordinates of the pixel
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	// To prevent trying to access data outside the image
	if (x >= cols || y >= rows)
		return;

	float c = 0.f; // This is a local variable that will hold the sum of pixel values of the neighbouring pixels

	for (int dx = -filterWidth / 2; dx < filterWidth / 2; dx++)
		// dx and dy represent the offset of the neighbouring pixels along the horizontal and vertical axes respectively corresponding to the anchor pixel
	{
		for (int dy = -filterWidth / 2; dy < filterWidth / 2; dy++)
		{
			// xx and yy represent the 2D coordinates of the neighbouring pixels
			int xx = min(max(x + dx, 0), cols - 1); //This takes care of the boundary conditions by extending the image 
			int yy = min(max(y + dy, 0), rows - 1);
			/*
			I have done this based on wikipedia page https://en.wikipedia.org/wiki/Kernel_(image_processing)#Edge_Handling which deals with image processing and  how edges were handled.
			I have followed the following approach here:
			The nearest border pixels are conceptually extended as far as necessary to provide values for the convolution.
			Corner pixels are extended in 90° wedges. Other edge pixels are extended in lines.
			*/
			c += inputChannel[yy * cols + xx];  // Image channels are 1D arrays, hence we need to offset the pixel coordinates to access the pixel in the channel array
		}
	}
	outputChannel[y * cols + x] = c / factor; // Same is the case here as well
}

void serial_box_blur(const Mat inputImage, Mat outputImage, int rows, int cols, int filterWidth, int factor, int channels)
{
	// For every pixel in the image
	for (int x = 0; x < cols; x++)
	{
		for (int y = 0; y < rows; y++)
		{
			float b = 0.f, g = 0.f, r = 0.f;
			// For every neighbouring pixel (based on the filter's width) around the pixel at (x,y)
			for (int dx = -filterWidth / 2; dx <= filterWidth / 2; dx++)
			{
				for (int dy = -filterWidth / 2; dy <= filterWidth / 2; dy++)
					// dx and dy represent the offset of the neighbouring pixels along the horizontal and vertical axes respectively corresponding to the anchor pixel
				{
					int yy = min(max(y + dy, 0), rows - 1);
					int xx = min(max(x + dx, 0), cols - 1);
					b += inputImage.data[channels * (cols * yy + xx) + 0];
					g += inputImage.data[channels * (cols * yy + xx) + 1];
					r += inputImage.data[channels * (cols * yy + xx) + 2];
				}
			}
			outputImage.data[channels * (cols * y + x) + 0] = b / factor;
			outputImage.data[channels * (cols * y + x) + 1] = g / factor;
			outputImage.data[channels * (cols * y + x) + 2] = r / factor;
		}
	}
}

__global__ void light_edge_detection(const unsigned char* inputChannel, unsigned char* outputChannel, int rows, int cols)
{
	// Calculating the coordinates of the pixel
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	// To prevent trying to access data outside the image
	if (x >= cols || y >= rows)
		return;

	float c = 0.f;
	int filter[] = { -1, -1, -1, -1, 8, -1, -1, -1, 0 };

	for (int dx = 0; dx < 9; dx++) // Here dx - 9/2  is the offset of the neighbouring pixels from the anchor pixel along the horizontal direction
	{
		int xx = x + dx - 9 / 2; // xx is the x coordinate of the neighbouring pixel
		xx = min(max(xx, 0), cols - 1); // Edge case consideration is same as that used for the box filter kernel
		c += (filter[dx] * inputChannel[y * cols + xx]);
	}
	// Again both above and below, image channels are 1D arrays, hence we need to offset the pixel coordinates to access the pixel in the channel array
	outputChannel[y * cols + x] = c;
}

void serial_light_edge_detection(const Mat inputImage, Mat outputImage, int rows, int cols, int channels)
{
	int filter[] = { -1, -1, -1, -1, 8, -1, -1, -1, 0 };
	// For every pixel in the image
	for (int x = 0; x < cols; x++)
	{
		for (int y = 0; y < rows; y++)
		{
			float b = 0.f, g = 0.f, r = 0.f;
			for (int dx = 0; dx < 9; dx++) // Here dx - 9/2  is the offset of the neighbouring pixels from the anchor pixel along the horizontal direction
			{
				int xx = x + dx - 9 / 2; // xx is the x coordinate of the neighbouring pixel
				xx = min(max(xx, 0), cols - 1); // Edge case consideration is same as that used for the box filter kernel
				b += filter[dx] * inputImage.data[channels * (cols * y + xx) + 0];
				g += filter[dx] * inputImage.data[channels * (cols * y + xx) + 1];
				r += filter[dx] * inputImage.data[channels * (cols * y + xx) + 2];
			}
			outputImage.data[channels * (cols * y + x) + 0] = b;
			outputImage.data[channels * (cols * y + x) + 1] = g;
			outputImage.data[channels * (cols * y + x) + 2] = r;
		}
	}
}

__global__ void separateChannels(const uchar4* inputImageRGBA, int rows, int cols, unsigned char* redChannel, unsigned char* greenChannel, unsigned char* blueChannel)
{
	// Calculating the coordinates of the pixel
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	// To prevent trying to access data outside the image
	if (x >= cols || y >= rows)
		return;

	int pixelPosition = y * cols + x; // Image channels are 1D arrays, hence we need to offset the pixel coordinates to access the pixel in the channel array

	redChannel[pixelPosition] = inputImageRGBA[pixelPosition].x;
	greenChannel[pixelPosition] = inputImageRGBA[pixelPosition].y;
	blueChannel[pixelPosition] = inputImageRGBA[pixelPosition].z;
}

__global__ void recombineChannels(const unsigned char* redChannel, const unsigned char* greenChannel, const unsigned char* blueChannel, uchar4* outputImageRGBA, int rows, int cols)
{
	// Calculating the coordinates of the pixel
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	// To prevent trying to access data outside the image
	if (x >= cols || y >= rows)
		return;

	int pixelPosition = y * cols + x; //Image channels are 1D arrays, hence we need to offset the pixel coordinates to access the pixel in the channel array

	unsigned char red = redChannel[pixelPosition];
	unsigned char green = greenChannel[pixelPosition];
	unsigned char blue = blueChannel[pixelPosition];

	// Alpha should be 255 for no transparency
	uchar4 outputPixel = make_uchar4(red, green, blue, 255); // This combines the red, green, blue and alpha channel values into a vector
	outputImageRGBA[pixelPosition] = outputPixel;
}

int main()
{
	char input_file[] = "1/Original.jpg";

	cv::Mat image = cv::imread(input_file, cv::IMREAD_COLOR);
	if (image.empty())
	{
		printf("Couldn't open the file %s\n", input_file);
		exit(1);
	}

	char output_file[] = "Images set 1/Blurred_GPU.jpg";
	char output_file2[] = "Images set 1/TotalVariationFilter_GPU.jpg";
	char output_file3[] = "Images set 1/Blurred_CPU.jpg";
	char output_file4[] = "Images set 1/TotalVariationFilter_CPU.jpg";

	int filterWidth = 9; // For the box blur
	int divFactor = filterWidth * filterWidth; // For dividing the sum of neighbouring pixel values after summation for the box filter for normalization

	// For all the variable names I have used the convention I learnt from the udacity course that h_ represents host (CPU) variable and d_ represents device (GPU) variable
	uchar4* h_inputImageRGBA, * d_inputImageRGBA;
	cv::Mat inputImageRGBA;

	// For box blur
	uchar4* d_outputImageRGBA;
	unsigned char* d_redBlurred, * d_greenBlurred, * d_blueBlurred;
	unsigned char* d_red, * d_green, * d_blue;
	cv::Mat outputImageRGBA;

	// For light edge
	uchar4* d_outputImageRGBA2;
	unsigned char* d_redlight, * d_greenlight, * d_bluelight;
	cv::Mat outputImageRGBA2; // Light edge filter application

	int cols = image.cols;
	int rows = image.rows;
	int totalPixels = cols * rows;
	int channels = image.channels();
	
	// For the serial code
	Mat CPUoutput1, CPUoutput2;
	CPUoutput1 = image.clone();
	CPUoutput2 = image.clone();

	clock_t startcpu, endcpu;
	double cpu_time_used;
	startcpu = clock();

	serial_box_blur(image, CPUoutput1, rows, cols, filterWidth, divFactor, channels);
	serial_light_edge_detection(image, CPUoutput2, rows, cols, channels);

	endcpu = clock();
	cpu_time_used = (((double)(endcpu - startcpu)) / CLOCKS_PER_SEC)*1000; // For milli seconds
	printf("Total time taken for both filters for image of size %d,%d on CPU: %lf ms\n", cols, rows, cpu_time_used);

	cv::imwrite(output_file3, CPUoutput1);
	cv::imwrite(output_file4, CPUoutput2);
	
	/*
	// I have compressed this 9 by 8 matrix into a 1D array and have used that instead of this
	int h_lightEdgeFilter[lightKernelWidth * lightKernelHeight] = { 0 }; // Initializing the light edge filter on the host
	for (int c = 0; c <= lightKernelWidth; c++)
		{
		for (int r = 0; r <= lightKernelHeight; r++)
			{
			if (r == 4) h_lightEdgeFilter[c * filterWidth + r] = 1;
			else if (r == c) h_lightEdgeFilter[c * filterWidth + r] = -1;
			}
		}
	*/

	cv::cvtColor(image, inputImageRGBA, cv::COLOR_BGR2BGRA);

	// Allocating memory for the outputs
	outputImageRGBA.create(rows, cols, CV_8UC4);
	outputImageRGBA2.create(rows, cols, CV_8UC4);

	h_inputImageRGBA = (uchar4*)inputImageRGBA.ptr<unsigned char>(0);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	hipStream_t s1, s2, s3, s4, s5, s6; // For parallelizing memory copies and kernel launches
	hipStreamCreate(&s1); hipStreamCreate(&s2); hipStreamCreate(&s3);
	hipStreamCreate(&s4); hipStreamCreate(&s5); hipStreamCreate(&s6);

	// Allotting memory for splitting the image into its different channels in GPU
	hipMalloc(&d_red, sizeof(unsigned char) * totalPixels);
	hipMalloc(&d_green, sizeof(unsigned char) * totalPixels);
	hipMalloc(&d_blue, sizeof(unsigned char) * totalPixels);

	// Alloting memory for the output images in the GPU
	hipMalloc(&d_inputImageRGBA, sizeof(uchar4) * totalPixels);
	hipMalloc(&d_outputImageRGBA, sizeof(uchar4) * totalPixels);
	hipMalloc(&d_outputImageRGBA2, sizeof(uchar4) * totalPixels);
	hipMemcpyAsync(d_inputImageRGBA, h_inputImageRGBA, sizeof(uchar4) * totalPixels, hipMemcpyHostToDevice, s1);

	// Alloting memory for each output channel on the GPU
	// For box blur
	hipMalloc(&d_redBlurred, sizeof(unsigned char) * totalPixels);
	hipMalloc(&d_greenBlurred, sizeof(unsigned char) * totalPixels);
	hipMalloc(&d_blueBlurred, sizeof(unsigned char) * totalPixels);

	// For light edge filter
	hipMalloc(&d_redlight, sizeof(unsigned char) * totalPixels);
	hipMalloc(&d_greenlight, sizeof(unsigned char) * totalPixels);
	hipMalloc(&d_bluelight, sizeof(unsigned char) * totalPixels);

	hipDeviceSynchronize();

	const dim3 blockSize(32, 32, 1);
	const dim3 gridSize((cols / blockSize.x) + 1, (rows / blockSize.y) + 1, 1);

	separateChannels <<<gridSize, blockSize>>> (d_inputImageRGBA, rows, cols, d_red, d_green, d_blue);
	hipDeviceSynchronize();


	//I have made processing of each channel to be run on different streams which gave me a significant speedup of 40% over running all on the same stream 
	box_blur <<<gridSize, blockSize, 0, s1>>> (d_red, d_redBlurred, rows, cols, filterWidth, divFactor);
	box_blur <<<gridSize, blockSize, 0, s2>>> (d_green, d_greenBlurred, rows, cols, filterWidth, divFactor);
	box_blur <<<gridSize, blockSize, 0, s3>>> (d_blue, d_blueBlurred, rows, cols, filterWidth, divFactor);

	light_edge_detection <<<gridSize, blockSize, 0, s4>>> (d_red, d_redlight, rows, cols);
	light_edge_detection <<<gridSize, blockSize, 0, s5>>> (d_green, d_greenlight, rows, cols);
	light_edge_detection <<<gridSize, blockSize, 0, s6>>> (d_blue, d_bluelight, rows, cols);

	hipDeviceSynchronize();

	recombineChannels <<<gridSize, blockSize, 0, s1>>> (d_redBlurred, d_greenBlurred, d_blueBlurred, d_outputImageRGBA, rows, cols);
	recombineChannels <<<gridSize, blockSize, 0, s2>>> (d_redlight, d_greenlight, d_bluelight, d_outputImageRGBA2, rows, cols);

	hipDeviceSynchronize();

	hipMemcpyAsync(outputImageRGBA.ptr<unsigned char>(0), d_outputImageRGBA, sizeof(uchar4) * totalPixels, hipMemcpyDeviceToHost, s1);
	hipMemcpyAsync(outputImageRGBA2.ptr<unsigned char>(0), d_outputImageRGBA2, sizeof(uchar4) * totalPixels, hipMemcpyDeviceToHost, s2);

	float milliseconds = 0;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("Total time taken for both filters for image of size %d,%d on GPU: %f ms\n", cols, rows, milliseconds);
	printf("The blur kernel used was %d,%d\n", filterWidth, filterWidth);

	cv::imwrite(output_file, outputImageRGBA);
	cv::imwrite(output_file2, outputImageRGBA2);

	printf("Process complete\n");

	hipFree(d_inputImageRGBA);	hipFree(d_outputImageRGBA);
	hipFree(d_redBlurred);	hipFree(d_red); hipFree(d_redlight);
	hipFree(d_greenBlurred); hipFree(d_green); hipFree(d_greenlight);
	hipFree(d_blueBlurred); hipFree(d_blue); hipFree(d_bluelight);
	hipStreamDestroy(s1); hipStreamDestroy(s2); hipStreamDestroy(s3); hipStreamDestroy(s4); hipStreamDestroy(s5); hipStreamDestroy(s6);

	return 0;
}

