#include "hip/hip_runtime.h"
#include<time.h>
#include <stdio.h>
#include<iostream>

#define N 10;

// CPU : 0.001s
// GPU : 0.00001 s

/*void knapSack(int value[], int weight[], int capacity, int n)
{
	//int dp[n + 1][capacity + 1];
	int* dp = (int*)malloc(sizeof(int)*(n+1)*(capacity+1));


	for (int i = 0; i <= capacity; i++)
		dp[i*(capacity+1)] = 0;
	for (int i = 0; i <= n; i++)
		dp[i] = 0;

	for (int i = 1; i <= n; i++)
	{
		for (int j = 1; j <= capacity; j++)
		{
			if (j >= weight[i - 1])
				dp[i*(capacity+1)+j] = dp[(i-1) * (capacity + 1) + j] < (value[i - 1] + dp[(i - 1)*(capacity+1) + j - weight[i - 1]]) ? (value[i - 1] + dp[(i - 1) * (capacity + 1) + j - weight[i - 1]]) : dp[(i - 1) * (capacity + 1) + j];
			else
				dp[i * (capacity + 1) + j] = dp[(i - 1) * (capacity + 1) + j];
			std::cout << dp[i * (capacity + 1) + j] << std::endl;
		}
	}
	std::cout << dp[capacity + n * (capacity + 1)] << std::endl;
	free(dp);dp = NULL;
}*/

void knapSack(int value[], int weight[], int capacity, int n)
{
	int** dp = new int* [n+1];
	for (int i = 0; i <=n; i++)
		dp[i] = new int[capacity+1];
	//int dp[n + 1][capacity + 1];

	for (int i = 0; i <= capacity; i++)
		dp[0][i] = 0;
	for (int i = 0; i <= n; i++)
		dp[i][0] = 0;

	for (int i = 1; i <= n; i++)
	{
		for (int j = 1; j <= capacity; j++)
		{
			if (j >= weight[i - 1])
				dp[i][j] = dp[i - 1][j] < (value[i - 1] + dp[i - 1][j - weight[i - 1]]) ? (value[i - 1] + dp[i - 1][j - weight[i - 1]]) : dp[i - 1][j];
			else
				dp[i][j] = dp[i - 1][j];
		}
	}
	/*DEBUGGING
	for (int i = 0; i <= n; i++)
	{
		for (int j = 0; j <= capacity; j++)
		{
			std::cout << dp[i][j] << " ";
		}
		std::cout << std::endl;
	}*/
	std::cout << dp[n][capacity] <<" is the maximum value from CPU\n" ;
	for (int i = 0; i <=n; i++)
		delete[] dp[i];
	delete[] dp;
}


// two types are shwon below

__global__ void knapsackGPU(int* dp, int row, int* d_value, int* d_weight,int capacity)
{
	int in = threadIdx.x + (blockDim.x * blockIdx.x);
	if (row != 0)
	{
		int ind = in + (row * (capacity+1));
		if (in <= (capacity+1) && in > 0)
		{
			if (in >= d_weight[row - 1])
			{
				dp[ind] = dp[ind - (capacity+1)]> (d_value[row - 1] + dp[ind - (capacity + 1) - d_weight[row - 1]]) ? dp[ind - (capacity + 1)] : (d_value[row - 1] + dp[ind - (capacity + 1) - d_weight[row - 1]]);
			}
			else
				dp[ind] = dp[ind - (capacity+1)];
		}
		if (in == 0)
		{
			dp[ind] = 0;
		}
	}
	else
	{
		dp[in] = 0;
	}
}


__global__ void knapsackGPU2(int* dp, int* d_value, int* d_weight, int capacity,int n)
{
	int in = threadIdx.x + (blockDim.x * blockIdx.x);
	for (int row = 0;row <= n;row++)
	{
		if (row != 0)
		{
			int ind = in + (row * (capacity + 1));
			if (in <= (capacity + 1) && in > 0)
			{
				if (in >= d_weight[row - 1])
				{
					dp[ind] = dp[ind - (capacity + 1)] > (d_value[row - 1] + dp[ind - (capacity + 1) - d_weight[row - 1]]) ? dp[ind - (capacity + 1)] : (d_value[row - 1] + dp[ind - (capacity + 1) - d_weight[row - 1]]);
				}
				else
					dp[ind] = dp[ind - (capacity + 1)];
			}
			if (in == 0)
			{
				dp[ind] = 0;
			}
		}
		else
		{
			dp[in] = 0;
		}
	}
	
}


int main()
{
	int val[] = { 60, 100, 120 };
	int wt[] = { 10, 20, 30 };
	int capacity = 50;
	int n = sizeof(val) / sizeof(val[0]);
	//knapSack(val, wt, capacity, n);

	int* d_value, * d_weight;
	hipMalloc((void**)&d_value, n * sizeof(int));
	hipMalloc((void**)&d_weight, n * sizeof(int));
	int* dp;
	hipMalloc((void**)&dp, (n + 1) * (capacity + 1) * sizeof(int));

	hipMemcpy(d_value, val, (n) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_weight, wt, (n) * sizeof(int), hipMemcpyHostToDevice);


	//dim3 block((capacity / N),1,1);
	//dim3 thread(N, 1, 1);
	int block = 1 + (capacity+1) / N;
	int thread = N;
	
	clock_t start, end;
	start = clock();
	knapSack(val, wt, capacity, n);
	end = clock();
	double time = ((double)end - (double)start) / CLOCKS_PER_SEC;

	printf("%f is the time taken by the CPU\n",time);



	start = clock();
	knapsackGPU2 <<<block,thread>>> (dp,d_value,d_weight,capacity,n);
	end = clock();
	time = ((double)end - (double)start) / CLOCKS_PER_SEC;

	printf("%f is the time taken by the GPU\n", time);
	// this was one method
	/*for (int i = 0;i <= n;i++)
	{
		knapsackGPU <<<block,thread>>> (dp, i, d_value, d_weight,capacity);
		cudaDeviceSynchronize();
		cudaError_t err = cudaGetLastError();
		if (err != cudaSuccess)
		{
			printf("CUDA Error: %s\n", cudaGetErrorString(err));
			exit(1);
		}
	}*/

	int *h_dp;
	h_dp = (int*)malloc(sizeof(int)*(n+1)*(capacity+1));
	hipMemcpy(h_dp, dp, (n + 1) * (capacity + 1) * sizeof(int), hipMemcpyDeviceToHost);

	printf("%d is the maximum value\n", h_dp[capacity + n*(capacity+1)]);
	
	/*for (int i = 0; i <= n; i++)
	{
		for (int j = 0; j <= capacity; j++)
		{
			std::cout << h_dp[j+i*(capacity+1)] << " ";
		}
		std::cout << std::endl;
	}*/

	return 0;
}
